
#include <hip/hip_runtime.h>

/*!
	Fills an array with N elements according to some divergent control flow
	with a loop.
*/
__global__ void k_sequenceLooping(float *ptr, int N) {
	int i = threadIdx.x;
	ptr[i] = cos((float)i);

}

/*!
	Computes

		R = A * V

	where A is an M x N column-major vector, V is an N element vector, and R is an M element vector.

	M and N are assumed to be multiples of block size which is 8
*/
__global__ void k_matrixVectorProduct(const float *A, const float *V, float *R, int M, int N) {
#define BLOCKSIZE 8
	
	__shared__ float V_buffer[BLOCKSIZE];

	A += blockIdx.x * blockDim.x + threadIdx.x;
	V += threadIdx.x;

	float r = 0;

	for (int i = 0; i < N; i += blockDim.x) {
		V_buffer[threadIdx.x] = *V;
		__syncthreads();

		for (int j = 0; j < blockDim.x; j++) {
			float a = *A;
			r += a * V_buffer[j];
			A += M;
		}
		V += blockDim.x;

		__syncthreads();
	}

	R[blockDim.x * blockIdx.x + threadIdx.x] = r;
}



