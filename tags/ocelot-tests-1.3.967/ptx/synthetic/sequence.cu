
#include <hip/hip_runtime.h>
/*!
	Defines a simple kernel
*/

__global__ void k_sequence(int *A, int N) {
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	if (n < N) {
		A[n] = 2*n+1;
	}
}

__global__ void k_simple_sequence(int *A) {
	int n = blockDim.x * blockIdx.x + threadIdx.x;
	A[n] = 2*n+1;
}

int main()
{
	k_sequence<<< 1, 1, 0 >>>( 0, 1);
}

