
#include <hip/hip_runtime.h>

__global__ void barrier( int* in, int* out )
{
	__shared__ int shared[8];
	shared[ ( threadIdx.x + 1 ) % blockDim.x ] = in[ threadIdx.x ];
	__syncthreads();
	out[ threadIdx.x ] = shared[ threadIdx.x ];
}

