#include "hip/hip_runtime.h"
/*!

	\file capModel3.cu
	
	\date Thursday December 18, 2008
	
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	
	\brief The source file for the CUDA CapModel3 simulation functions

*/

#ifndef CAP_MODEL_3_CU_INCLUDED
#define CAP_MODEL_3_CU_INCLUDED

#include "capModel3.hu"
#include "Cuda.h"
#include "CudaException.h"
#include "macros.h"
#include <stdint.h>

#define MAX_THREADS 64
#define MAX_CTAS 65535

////////////////////////////////////////////////////////////////////////////////
// Mersenne twister

static __device__ float mersenneTwister( CudaMersenneTwisterData* data )
{

	int iState, iState1, iStateM;
	unsigned int mti, mti1, mtiM, x;
	unsigned int mt[MT_NN];

	//Load bit-vector Mersenne Twister parameters
	CudaMersenneTwisterData config = *data;
	data->seed++;

	//Initialize current state
	mt[0] = config.seed;
	for(iState = 1; iState < MT_NN; iState++)
	{

		mt[iState] = (1812433253U * (mt[iState - 1] ^ 
			(mt[iState - 1] >> 30)) + iState) & MT_WMASK;

	}

	iState = 0;
	mti1 = mt[0];		

	iState1 = iState + 1;
	iStateM = iState + MT_MM;

	if(iState1 >= MT_NN)
	{
	
		iState1 -= MT_NN;
	
	}
	
	if(iStateM >= MT_NN)
	{
	
		iStateM -= MT_NN;
	
	}

	mti  = mti1;
	mti1 = mt[iState1];
	mtiM = mt[iStateM];

	x    = (mti & MT_UMASK) | (mti1 & MT_LMASK);
	x    =  mtiM ^ (x >> 1) ^ ((x & 1) ? config.matrix_a : 0);
	mt[iState] = x;
	iState = iState1;

	//Tempering transformation
	x ^= (x >> MT_SHIFT0);
	x ^= (x << MT_SHIFTB) & config.mask_b;
	x ^= (x << MT_SHIFTC) & config.mask_c;
	x ^= (x >> MT_SHIFT1);

	//Convert to (0, 1] float and write to global memory
	return ((float)x + 1.0f) / 4294967296.0f;	
	
}

////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
// Random Number Generation

__device__ __constant__ FLOAT wtab[128];
__device__ __constant__ unsigned long ktab[128];
__device__ __constant__ FLOAT ytab[128];

static __device__  FLOAT get_del( FLOAT x, FLOAT rational)
{

    FLOAT xsq = 0.0;
    FLOAT del = 0.0;
    FLOAT result = 0.0;

    xsq = floor (x * GAUSS_SCALE) / GAUSS_SCALE;
    del = (x - xsq) * (x + xsq);
    del *= 0.5;

    result = exp (-0.5 * xsq * xsq) * exp (-1.0 * del) * rational;

    return result;

}

static __device__ FLOAT gauss_small (const FLOAT x)
{

    unsigned int i;
    FLOAT result = 0.0;
    FLOAT xsq;
    FLOAT xnum;
    FLOAT xden;

    const FLOAT a[5] = 
    {
        2.2352520354606839287,
        161.02823106855587881,
        1067.6894854603709582,
        18154.981253343561249,
        0.065682337918207449113
    };
    
    const FLOAT b[4] = 
    {
        47.20258190468824187,
        976.09855173777669322,
        10260.932208618978205,
        45507.789335026729956
    };

    xsq = x * x;
    xnum = a[4] * xsq;
    xden = xsq;

    for (i = 0; i < 3; i++)
    {

        xnum = (xnum + a[i]) * xsq;
        xden = (xden + b[i]) * xsq;

    }

    result = x * (xnum + a[3]) / (xden + b[3]);

    return result;

}

static __device__ FLOAT gauss_medium (const FLOAT x)
{

    unsigned int i;
    FLOAT temp = 0.0;
    FLOAT result = 0.0;
    FLOAT xnum;
    FLOAT xden;
    FLOAT absx;

    const FLOAT c[9] = 
    {
        0.39894151208813466764,
        8.8831497943883759412,
        93.506656132177855979,
        597.27027639480026226,
        2494.5375852903726711,
        6848.1904505362823326,
        11602.651437647350124,
        9842.7148383839780218,
        1.0765576773720192317e-8
    };
    
    const FLOAT d[8] = 
    {
        22.266688044328115691,
        235.38790178262499861,
        1519.377599407554805,
        6485.558298266760755,
        18615.571640885098091,
        34900.952721145977266,
        38912.003286093271411,
        19685.429676859990727
    };

    absx = fabs (x);

    xnum = c[8] * absx;
    xden = absx;

    for (i = 0; i < 7; i++)
    {

        xnum = (xnum + c[i]) * absx;
        xden = (xden + d[i]) * absx;

    }

    temp = (xnum + c[7]) / (xden + d[7]);

    result = get_del (x, temp);

    return result;

}

static __device__ FLOAT gauss_large (const FLOAT x)
{
    
    int i;
    FLOAT result;
    FLOAT xsq;
    FLOAT temp;
    FLOAT xnum;
    FLOAT xden;
    FLOAT absx;

    const FLOAT p[6] = 
    {
        0.21589853405795699,
        0.1274011611602473639,
        0.022235277870649807,
        0.001421619193227893466,
        2.9112874951168792e-5,
        0.02307344176494017303
    };
    const FLOAT q[5] = 
    {
        1.28426009614491121,
        0.468238212480865118,
        0.0659881378689285515,
        0.00378239633202758244,
        7.29751555083966205e-5
    };

    absx = fabs (x);
    xsq = 1.0 / (x * x);
    xnum = p[5] * xsq;
    xden = xsq;

    for (i = 0; i < 4; i++)
    {

        xnum = (xnum + p[i]) * xsq;
        xden = (xden + q[i]) * xsq;

    }

    temp = xsq * (xnum + p[4]) / (xden + q[4]);
    temp = (M_1_SQRT2PI - temp) / absx;

    result = get_del (x, temp);

    return result;
}

static __device__ FLOAT normalCdfSingle( FLOAT X, FLOAT u, FLOAT sigma )
{

    X = ( X - u ) / sigma;

    FLOAT result;
    FLOAT absx = fabs (X);

    if (absx < GAUSS_EPSILON)
    {
        
        result = .5;

    }
    else if (absx < 0.66291)
    {

        result = 0.5 + gauss_small (X);

    }
    else if (absx < SQRT32)
    {

        result = gauss_medium (X);

        if (X > 0.0)
        {

            result = 1.0 - result;

        }


    }
    else if (X > GAUSS_XUPPER)
    {

        result = 1.0;

    }
    else if (X < GAUSS_XLOWER)
    {

        result = 0.0;

    }
    else
    {

        result = gauss_large (X);

        if (X > 0.0)
        {

            result = 1.0 - result;

        }

    }

    return result;

}

static __device__ FLOAT normalSample( CudaMersenneTwisterData* data, 
	const FLOAT sigma )
{

	FLOAT x;
	FLOAT y;
	FLOAT rSquared;
	
	do
	{
	
		x = -1.0 + 2.0 * mersenneTwister(data);
		y = -1.0 + 2.0 * mersenneTwister(data);
		
		rSquared = x * x + y * y;
	
	}
	while( rSquared > 1.0 || rSquared == 0 );
	
	return sigma * y * sqrt( -2.0 * log( rSquared )/ rSquared );

}

__device__ void multivariateNormal( CudaMersenneTwisterData* data, 
	FLOAT* samples, const FLOAT* mean, FLOAT* cov, unsigned int N, 
	FLOAT* random )
{

	for( unsigned int i = 0; i < N; ++i )
	{
	
		random[i] = normalSample( data, 1 );			
			
	}

	for( unsigned int i = 0; i < N; ++i )
	{
	
		samples[i] = mean[i];
		FLOAT total = 0;
		
		for( unsigned int j = 0; j < N; ++j )
		{
		
			samples[i] += cov[ j * N + i ] * random[i];
			total += cov[ j * N + i ];
		
		}
		
		samples[i] /= total;		
	
	}
	
}

__device__ void normalCdf( CudaMersenneTwisterData* data, FLOAT* samples, 
	FLOAT mean, FLOAT variance, unsigned int N )
{

	int i;
	
	for( i = 0; i < N; ++i )
	{
	
		samples[i] = normalCdfSingle( samples[i], mean, variance );
	
	}

}

static __device__ FLOAT gaussianZiggurat( CudaMersenneTwisterData* data, 
	FLOAT sigma )
{

	unsigned long int i, j;
	int sign;
	FLOAT x, y;

	while(1)
	{
		
		i = mersenneTwister(data) * 256; 
		j = mersenneTwister(data) * 16777216;
		sign = (i & 0x80) ? +1 : -1;
		i &= 0x7f;

		x = j * wtab[i];

		if ( j < ktab[i] )
		{
		
			break;

		}

		if (i < 127)
		{
		
			FLOAT y0, y1, U1;
			y0 = ytab[i];
			y1 = ytab[i + 1];
			U1 = mersenneTwister(data);
			y = y1 + (y0 - y1) * U1;
			
		}
		else
		{
		
			double U1, U2;
			U1 = 1.0 - mersenneTwister(data);
			U2 = mersenneTwister(data);
			x = PARAM_R - log (U1) / PARAM_R;
			y = exp (-PARAM_R * (x - 0.5 * PARAM_R)) * U2;
		
		}

		if (y < exp (-0.5 * x * x))
		{
			break;
		
		}
		
	}

	return sign * sigma * x;

}

static __device__ FLOAT gammaBase( CudaMersenneTwisterData* data, FLOAT a, 
	FLOAT b )
{

	FLOAT x, v, u;
	FLOAT d = a - 1.0 / 3.0;
	FLOAT c = ( 1.0 / 3.0 ) / sqrt( d );
	
	while( 1 )
	{
	
		do
		{
		
			x = gaussianZiggurat( data, 1.0 );
			v = 1.0 + c * x;
		
		}
		while( v <= 0 );
		
		v = v * v * v;
		
		u = mersenneTwister(data);
		
		if( u < 1 - 0.0331 * x * x * x * x )
		{
		
			break;
		
		}
		
		if( log(u) < 0.5 * x * x + d * ( 1 - v + log ( v ) ) )
		{
		
			break;
		
		}
	
	}
	
	return b * d * v;

}

static __device__ FLOAT gamma( CudaMersenneTwisterData* data, FLOAT a, FLOAT b )
{

	FLOAT factor = 1.0;

	while( a < 1 )
	{
	
		FLOAT u = mersenneTwister(data);
		a += 1.0;
		factor *= pow( u, (float) ( 1.0 / a ) );
	
	}
	
	return gammaBase( data, a, b ) * factor;

}

static __device__ FLOAT beta( CudaMersenneTwisterData* data, FLOAT a, FLOAT b )
{

	FLOAT x1 = gamma( data, a, 1.0 );
	FLOAT x2 = gamma( data, b, 1.0 );
	
	return x1 / ( x1 + x2 );

}

static __device__ FLOAT betaDistributionFromNormal( 
	CudaMersenneTwisterData* data, FLOAT mean, FLOAT stdev )
{

	FLOAT y = stdev / mean;
	FLOAT y2 = y * y;
	
	FLOAT alpha = ( 1.0 - mean - mean * y2 ) / y2;
	FLOAT b = alpha * ( 1.0 / mean - 1.0 );

	return beta( data, alpha, b );

}
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
// DEVICE FUNCTIONS
static __host__ __device__ unsigned int scratchSize( CudaInvariants* 
	invariants )
{

	// scratch object
	unsigned int size = sizeof( CudaScratch );

	// samples
	size += invariants->parameters->chunkSize * invariants->obligorSize * 
		sizeof( FLOAT );

	// counter cf
	size += ( 3 * sizeof( FLOAT ) * invariants->quarterSize + 
		sizeof( CudaCounterCf ) ) * invariants->counterPartySize;

	//cdsIncomeCf;
	//cdsExpenseCf;
	//marketPrice;
	//coupons;
	//discount;
	//capitalBalance;
	//premium;
	//investIncrement;
	//income;
	//claims;
	//paid;
	//operatingExpense;
	//debtService;
	//managementFee;
	//expense;
	//loss;
	//debtLoss;
	
	size += sizeof( FLOAT ) * invariants->quarterSize * 17;
	
	// random
	size += invariants->obligorSize * sizeof( FLOAT );
	
	return size;

}

static __host__ __device__ unsigned int invariantSize( CudaInvariants* 
	invariants, unsigned int threads )
{

	unsigned int size = sizeof( CudaInvariants );
	
	size += threads * sizeof( CudaMersenneTwisterData );
	size += sizeof( CudaSimulationParameters );
	size += invariants->obligorSize * sizeof( CudaObligor );
	size += invariants->portfolioSize * sizeof( CudaPortfolio );
	size += invariants->quarterSize * sizeof( unsigned int );
	
	size += invariants->obligorSize * sizeof( FLOAT );
	size += invariants->obligorSize * invariants->obligorSize * sizeof( FLOAT );
	size += invariants->obligorSize * sizeof( FLOAT );

	return size;

}

static __device__ void initializeScratch( CudaInvariants* 
	invariants, CudaScratch* scratch )
{

	uint8_t* data = (uint8_t*) scratch;

	// samples
	data += sizeof( CudaScratch );
	scratch->samples = ( FLOAT* ) ( data );

	//countercf
	data += invariants->parameters->chunkSize * invariants->obligorSize * 
		sizeof( FLOAT );
	scratch->countercf = ( CudaCounterCf* ) data;
	
	data += invariants->counterPartySize * sizeof( CudaCounterCf );
	
	for( unsigned int i = 0; i < invariants->counterPartySize; ++i )
	{
	
		// income
		scratch->countercf[i].income = (FLOAT*) data;
		
		// expense
		data += sizeof( FLOAT ) * invariants->quarterSize;
		scratch->countercf[i].expense = (FLOAT*) data;
		
		// loss
		data += sizeof( FLOAT ) * invariants->quarterSize;
		scratch->countercf[i].loss = (FLOAT*) data;
		
		data += sizeof( FLOAT ) * invariants->quarterSize;
	
	}

	//cdsIncomeCf;
	scratch->cdsIncomeCf = (FLOAT*) data;

	//cdsExpenseCf;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->cdsExpenseCf = (FLOAT*) data;

	//marketPrice;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->marketPrice = (FLOAT*) data;

	//coupons;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->coupons = (FLOAT*) data;

	//discount;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->discount = (FLOAT*) data;

	//capitalBalance;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->capitalBalance = (FLOAT*) data;

	//premium;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->premium = (FLOAT*) data;

	//investIncrement;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->investIncrement = (FLOAT*) data;

	//income;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->income = (FLOAT*) data;

	//claims;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->claims = (FLOAT*) data;

	//paid;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->paid = (FLOAT*) data;

	//operatingExpense;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->operatingExpense = (FLOAT*) data;

	//debtService;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->debtService = (FLOAT*) data;

	//managementFee;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->managementFee = (FLOAT*) data;

	//expense;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->expense = (FLOAT*) data;

	//loss;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->loss = (FLOAT*) data;

	//debtLoss;
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->debtLoss = (FLOAT*) data;
	
	// random
	data += sizeof( FLOAT ) * invariants->quarterSize;
	scratch->random = (FLOAT*) data;

}

static __device__ void initializeInvariants( CudaInvariants* invariants )
{

	uint8_t* data = ( uint8_t* ) invariants;
	
	data += sizeof( CudaInvariants );
	invariants->twisters = ( CudaMersenneTwisterData* ) data;
	
	data += TOTAL_THREADS() * sizeof( CudaMersenneTwisterData );
	invariants->parameters = ( CudaSimulationParameters* ) data;
	
	data += sizeof( CudaSimulationParameters );
	invariants->obligors = ( CudaObligor* ) data;
	
	data += invariants->obligorSize * sizeof( CudaObligor );
	invariants->portfolios = ( CudaPortfolio* ) data;
	
	data += invariants->portfolioSize * sizeof( CudaPortfolio );
	invariants->quarters = ( unsigned int* ) data;
	
	data += invariants->quarterSize * sizeof( unsigned int );
	invariants->mean = ( FLOAT* ) data;
	
	
	data += invariants->obligorSize * sizeof( FLOAT );
	invariants->correlation = ( FLOAT* ) data;
	
	data += invariants->obligorSize * invariants->obligorSize * sizeof( FLOAT );
	invariants->defaultProbability = ( FLOAT* ) data;
	
}

static __device__ void clearResult( CudaResult* result )
{

	result->claimProbability = 0.0;
	result->defaultProbability = 0.0;
	result->expectedClaim = 0.0;
	result->expectedLoss = 0.0;
	result->expectedLossPercent = 0.0;

	result->maxClaim = 0.0;
	result->maxClaimPv = 0.0;

}

static __device__ void capModel3Thread( void* shared, CudaInvariants* 
	invariants, CudaScratch* scratch, CudaResult* results )
{

	// initialize scratch
	initializeScratch( invariants, scratch );

	// clear results
	for( unsigned int i = 0; i < ( invariants->counterPartySize + 1); ++i )
	{
	
		clearResult( results + i );
	
	}
	
	// step 2 - generate random numbers
	
	// set seed
	invariants->twisters[ GLOBAL_ID() ].seed = invariants->seed;
	
	// assume the correlation has already been passed through the 
	// cholesky decomposition
	
	for( unsigned int i = 0; i < invariants->parameters->chunkSize; ++i )
	{
	
		multivariateNormal( invariants->twisters + GLOBAL_ID(), 
			scratch->samples + ( i * invariants->obligorSize ), 
			invariants->mean, invariants->correlation, 
			invariants->obligorSize, scratch->random );			
		normalCdf( invariants->twisters + GLOBAL_ID(),
			scratch->samples + ( i * invariants->obligorSize ), 0, 1, 
			invariants->obligorSize );
	
	}
	
	for( unsigned int i = 0; i < invariants->parameters->chunkSize; ++i )
	{
	
		for( unsigned int j = 0; j < invariants->obligorSize; ++j )
		{

			scratch->samples[i * invariants->obligorSize + j] = 10.0 * log( 
				scratch->samples[i * invariants->obligorSize + j] ) / 
				log( 1.0 - invariants->defaultProbability[j] );
			
		}
	
	}
	
	// main simulation loop
	for( unsigned int simNumber = 0; simNumber < 
		invariants->parameters->chunkSize; 
		++simNumber )
	{
	
		// step 3
	
		// zero out counter parties
		for( unsigned int i = 0; i < invariants->counterPartySize; ++i )
		{
		
			for( unsigned int j = 0; j < invariants->quarterSize; ++j )
			{
			
				scratch->countercf[i].income[j] = 0.0;
				scratch->countercf[i].expense[j] = 0.0;
				scratch->countercf[i].loss[j] = 0.0;
			
			}
		
		}
		
		// for each portfolio
		for( unsigned int portfolioId = 0; 
			portfolioId < invariants->portfolioSize; ++portfolioId )
		{
		
			// zero out cds stats
			for( unsigned int j = 0; j < invariants->quarterSize; ++j )
			{
			
				scratch->cdsIncomeCf[j] = 0.0;
				scratch->cdsExpenseCf[j] = 0.0;
			
			}
			
			const CudaPortfolio* portfolio = 
				invariants->portfolios + portfolioId;
			
			for( unsigned int i = 0; 
				i < MIN( portfolio->quartersToMaturity + 1, 
				invariants->quarterSize ); ++i )
			{
			
				scratch->cdsIncomeCf[i] = portfolio->quantity * 
					portfolio->spread / 4.0;
			
			}
			
			FLOAT value = scratch->samples[ simNumber * 
				invariants->obligorSize + portfolio->obligorId ];
				
			if( value < portfolio->yearsToMaturity )
			{
			
				unsigned int defaultQuarter = 0;
				bool found = false;
					
				for( ; defaultQuarter < invariants->quarterSize - 1; 
					++defaultQuarter )
				{
				
					if( (invariants->quarters)[defaultQuarter + 1] > value )
					{
					
						found = true;
						break;
					
					}
				
				}
						
				if( !found )
				{
		
					++defaultQuarter;
		
				}
				
				for( unsigned int i = defaultQuarter; 
					i < invariants->quarterSize; ++i )
				{
				
					scratch->cdsIncomeCf[i] = 0.0;
				
				}
				
				scratch->cdsExpenseCf[ defaultQuarter ] = 
					portfolio->quantity * 
					betaDistributionFromNormal( 
					invariants->twisters + GLOBAL_ID(), 
					invariants->parameters->lgdMean, 
					invariants->parameters->lgdStdev );
			
			}
			
			__syncthreads();
			
			CudaCounterCf* counter = scratch->countercf + portfolio->counterId;
			
			if( portfolio->quantity > 0 )
			{
				
				for( unsigned int i = 0; i < invariants->quarterSize; ++i )
				{
				
					counter->income[i] += scratch->cdsIncomeCf[i];
					counter->expense[i] += scratch->cdsExpenseCf[i];
				
				}
			
			}
			else
			{
			
				for( unsigned int i = 0; i < invariants->quarterSize; ++i )
				{
				
					counter->income[i] -= scratch->cdsExpenseCf[i];
					counter->expense[i] -= scratch->cdsIncomeCf[i];
				
				}
			
			}
		
		}
			
		// iterate over all counters
		for( unsigned int counterId = 0; 
			counterId < invariants->counterPartySize; ++counterId )
		{
		
			unsigned int defaultQuarter = 0;
			bool found = false;
			FLOAT value = scratch->samples[ simNumber * 
				invariants->obligorSize + counterId ];
				
			for( ; defaultQuarter < invariants->quarterSize - 1; 
				++defaultQuarter )
			{
			
				if( (invariants->quarters)[defaultQuarter + 1] > value )
				{
				
					found = true;
					break;
				
				}
			
			}
					
			if( !found )
			{
	
				++defaultQuarter;
	
			}
			
			for( unsigned int i = defaultQuarter; 
				i < invariants->quarterSize; ++i )
			{
			
				scratch->countercf[counterId].income[i] = 0.0;
			
			}
		
		}
			
		__syncthreads();
		
		//step 4
		for( unsigned int j = 0; j < invariants->quarterSize; ++j )
		{
		
			scratch->marketPrice[j] = 1.0;
			scratch->coupons[j] = invariants->parameters->investRate / 4.0;
			scratch->discount[j] = ( 1.0 / pow( (float) ( 1 + 
				invariants->parameters->investRate / 4.0 ), 
				(float)( j + 1 ) ) );
		
		}
		
		// step 5
		FLOAT capital = invariants->parameters->initialEquity + 
			invariants->parameters->debtParameter;
		
		for( unsigned int j = 0; j < invariants->quarterSize; ++j )
		{
		
			scratch->capitalBalance[j] = 0.0;
			scratch->premium[j] = 0.0;
			scratch->investIncrement[j] = 0.0;

			scratch->income[j] = 0.0;
			scratch->claims[j] = 0.0;
			scratch->paid[j] = 0.0;

			scratch->debtService[j] = 0.0;
			scratch->expense[j] = 0.0;
			scratch->loss[j] = 0.0;

			scratch->debtLoss[j] = 0.0;
			scratch->managementFee[j] = 0.0;
			
			scratch->operatingExpense[j] = 
				( invariants->parameters->optionExpense / 4.0 ) * 
				pow( (float) ( 1.0 + invariants->parameters->optionInflation 
				/ 4.0 ) , (float) (j + 1) );
		
		}
		
		bool inRunOff = false;
		
		for( unsigned int quarter = 0; 
			quarter < invariants->quarterSize; ++quarter )
		{
		
			// premium and recoveries
			for( unsigned int i = 0; i < invariants->counterPartySize; ++i )
			{
		
				scratch->premium[ quarter ] += 
					scratch->countercf[i].income[ quarter ];
		
			}
			
			// investment income
			scratch->investIncrement[ quarter ] = MAX( 0.0, capital * 
				scratch->coupons[ quarter ] / scratch->marketPrice[ quarter ] );
				
			// total income
			scratch->income[ quarter ] += scratch->premium[ quarter ] + 
				scratch->investIncrement[ quarter ];
			capital += scratch->income[ quarter ];
		
			// 1st claims
			for( unsigned int i = 0; i < invariants->counterPartySize; ++i )
			{
		
				scratch->claims[ quarter ] += 
					scratch->countercf[i].expense[ quarter ];
		
			}
			
			scratch->paid[ quarter ] = MIN( capital, 
				scratch->claims[ quarter ] );
			capital -= scratch->paid[ quarter ];
			scratch->expense[ quarter ] += scratch->paid[ quarter ];
			
			if( scratch->paid[ quarter ] < scratch->claims[ quarter ] )
			{
			
				inRunOff = true;
				
				for( unsigned int j = 0; j < invariants->counterPartySize; ++j )
				{
		
					if( scratch->countercf[j].expense[ quarter ] > 0 )
					{
					
						for( unsigned int i = quarter + 1; 
							i < invariants->quarterSize; ++i )
						{
						
							scratch->countercf[j].income[i] = 0;
						
						}
					
					}
		
				}
				
				scratch->loss[ quarter ] = scratch->claims[ quarter ] - 
					scratch->paid[ quarter ];
				FLOAT percentPaid = scratch->paid[ quarter ] / 
					scratch->claims[ quarter ];
			
				for( unsigned int j = 0; j < invariants->counterPartySize; ++j )
				{
				
					scratch->countercf[j].loss[ quarter ] = 
						scratch->countercf[j].expense[ quarter ] * 
						( 1.0 - percentPaid );
				
				}
			
			}
			
			__syncthreads();
			
			// debt service
			scratch->debtService[ quarter ] = 
				invariants->parameters->debtParameter * 
				invariants->parameters->debtRate;
			scratch->debtService[ quarter ] = MAX( 0, 
				MIN( scratch->debtService[ quarter ], capital ) );
			capital -= scratch->debtService[ quarter ];
			scratch->expense[ quarter ] += scratch->debtService[ quarter ];
			
			if( capital < invariants->parameters->debtParameter )
			{
			
				scratch->debtLoss[quarter] = MAX( 0, 
					invariants->parameters->debtParameter - capital );
			
			}
			
			__syncthreads();
			
			// operating expenses
			scratch->operatingExpense[ quarter ] = 
				MIN( scratch->operatingExpense[ quarter ], MAX( capital, 0 ) );
			capital -= scratch->operatingExpense[ quarter ];
			scratch->expense[ quarter ] += scratch->operatingExpense[ quarter ];
			
			// management fee
			if( !inRunOff )
			{
			
				FLOAT assetBase;
			
				if( quarter == 0 )
				{
				
					assetBase = invariants->parameters->initialEquity + 
						invariants->parameters->debtParameter;
				
				}
				else
				{
				
					assetBase = scratch->capitalBalance[ quarter - 1 ];
				
				}
				
				FLOAT assetFee = 0.25 * 
					invariants->parameters->assetManagementFeeRate * assetBase;
					
				FLOAT sum = 0;
				
				for( unsigned int portfolioId = 0; 
					portfolioId < invariants->portfolioSize; ++portfolioId )
				{
				
					sum += invariants->portfolios[portfolioId].quantity;
				
				}
				
				scratch->managementFee[quarter] = 
					MIN( MAX( capital, 0 ), assetFee + 
					MIN( 0.25 * invariants->parameters->managementFeeRate 
					* sum, 0.25 * invariants->parameters->managementFeeCap ) );
									
			}
			
			__syncthreads();
			
			capital -= scratch->managementFee[ quarter ];
			scratch->expense[ quarter ] += scratch->managementFee[ quarter ];
			scratch->capitalBalance[ quarter ] = capital;
			capital = MAX( capital, 0 );
		
		}
		
		// step 6
		
		for( unsigned int j = 0; j < invariants->counterPartySize; ++j )
		{
		
			CudaCounterCf* v = scratch->countercf + j;
			CudaResult* r = results + j;
			
			FLOAT pvClaim = 0;
			FLOAT pvLoss = 0;
			FLOAT cpClaim = 0;
			FLOAT cpLoss = 0;
			
			for( unsigned int i = 0; i < invariants->quarterSize; ++i )
			{
			
				pvClaim += v->expense[i] * scratch->discount[i];
				pvLoss += v->loss[i] * scratch->discount[i];
				cpClaim += v->expense[i];
				cpLoss += v->loss[i];
			
			}
							
			if( pvClaim > 0 )
			{
			
				
				r->claimProbability += 1;
				r->expectedClaim += pvClaim;
				r->maxClaim = MAX( r->maxClaim, cpClaim );
				r->maxClaimPv = MAX( r->maxClaimPv, pvClaim );
				
				if( pvLoss > 0 )
				{
				
					r->defaultProbability += 1;
					r->expectedClaim += pvLoss;
					r->expectedLossPercent += ( pvLoss / pvClaim );
				
				}
				
			}
			
			__syncthreads();
		
		}
		
		FLOAT pvClaim = 0;
		FLOAT pvLoss = 0;
		FLOAT cpClaim = 0;
		FLOAT cpLoss = 0;
		
		for( unsigned int i = 0; i < invariants->quarterSize; ++i )
		{
		
			pvClaim += scratch->claims[i] * scratch->discount[i];
			pvLoss += scratch->loss[i] * scratch->discount[i];
			cpClaim += scratch->claims[i];
			cpLoss += scratch->loss[i];
		
		}
		
		CudaResult* r = &results[invariants->counterPartySize];
		
		if( pvClaim > 0 )
		{
		
			r->claimProbability += 1;
			r->expectedClaim += pvClaim;
			r->maxClaim = MAX( r->maxClaim, cpClaim );
			r->maxClaimPv = MAX( r->maxClaimPv, pvClaim );
			
			if( pvLoss > 0 )
			{
			
				r->defaultProbability += 1;
				r->expectedClaim += pvLoss;
				r->expectedLossPercent += ( pvLoss / pvClaim );
			
			}
			
		}
		
		__syncthreads();
	
	}

}
////////////////////////////////////////////////////////////////////////////////

////////////////////////////////////////////////////////////////////////////////
// GLOBAL FUNCTIONS
__global__ void capModel3Kernel( CudaInvariants* invariants, 
	CudaScratch* scratch, CudaResult* results )
{

	unsigned int id = GLOBAL_ID();
	
	extern __shared__ int shared[];
	
	// initialize invariants
	// this is not normally thread safe, but okay since every grid 
	// writes the same data
	if( THREAD_ID() == 0 )
	{
	
		initializeInvariants( invariants );
	
	}
	
	__syncthreads();
	
	uint8_t* scratchData = (uint8_t*) scratch;
	unsigned int scratchBytes = scratchSize( invariants );
	CudaScratch* scratchPointer = ( CudaScratch* ) ( scratchData + 
		IMUL( scratchBytes, id ) );
		
	uint8_t* resultData = (uint8_t*) results;
	unsigned int resultBytes = sizeof( CudaResult ) * 
		( invariants->counterPartySize + 1);
	CudaResult* resultPointer = ( CudaResult* ) ( IMUL( id, resultBytes ) + 
		resultData );
	
	capModel3Thread( (void*)shared, invariants, scratchPointer, resultPointer );
	
}

////////////////////////////////////////////////////////////////////////////////


////////////////////////////////////////////////////////////////////////////////
// C FUNCTIONS

static void packInvariants( CudaInvariants* result, 
	const CudaInvariants* source, unsigned int threads )
{

	uint8_t* data = ( uint8_t* ) result;
	
	data += sizeof( CudaInvariants );
	memcpy( data, source->twisters, threads * 
		sizeof( CudaMersenneTwisterData ) );
	result->twisters = ( CudaMersenneTwisterData* ) data;
	
	data += threads * sizeof( CudaMersenneTwisterData );
	memcpy( data, source->parameters, sizeof( CudaSimulationParameters ) );
	result->parameters = ( CudaSimulationParameters* ) data;
	
	data += sizeof( CudaSimulationParameters );
	memcpy( data, source->obligors, 
		source->obligorSize * sizeof( CudaObligor ) );
	result->obligors = ( CudaObligor* ) data;
	
	data += source->obligorSize * sizeof( CudaObligor );
	memcpy( data, source->portfolios, 
		source->portfolioSize * sizeof( CudaPortfolio ) );
	result->portfolios = ( CudaPortfolio* ) data;
	
	data += source->portfolioSize * sizeof( CudaPortfolio );
	memcpy( data, source->quarters, 
		source->quarterSize * sizeof( unsigned int ) );
	result->quarters = ( unsigned int* ) data;
	
	data += source->quarterSize * sizeof( unsigned int );
	memcpy( data, source->mean, 
		source->obligorSize * sizeof( FLOAT ) );
	result->mean = ( FLOAT* ) data;
	
	data += source->obligorSize * sizeof( FLOAT );
	memcpy( data, source->correlation, 
		source->obligorSize * source->obligorSize * sizeof( FLOAT ) );
	result->correlation = ( FLOAT* ) data;
	
	data += source->obligorSize * source->obligorSize * sizeof( FLOAT );
	memcpy( data, source->defaultProbability, 
		source->obligorSize * sizeof( FLOAT ) );
	result->defaultProbability = ( FLOAT* ) data;
		
	// size parameters
	result->device = source->device;
	result->seed = source->seed;
	
	result->maxTwisters = source->maxTwisters;
	
	result->obligorSize = source->obligorSize;
	result->portfolioSize = source->portfolioSize;
	
	result->counterPartySize = source->counterPartySize;
	result->quarterSize = source->quarterSize;
	
}

static void reduceResults( CudaResult* results, CudaResult* expandedResults, 
	CudaInvariants* invariants, unsigned int threads, FLOAT correction )
{

	unsigned int step = ( invariants->counterPartySize + 1);

	// clear results
	for( unsigned int j = 0; j < step; ++j )
	{
	
		results[j].claimProbability = 0.0;
		results[j].defaultProbability = 0.0;
		results[j].expectedClaim = 0.0;
		results[j].expectedLoss = 0.0;
		results[j].expectedLossPercent = 0.0;

		results[j].maxClaim = 0.0;
		results[j].maxClaimPv = 0.0;
	
	}

	// reduce the expanded results
	for( unsigned int i = 0; i < threads; ++i )
	{
	
		for( unsigned int j = 0; j < step; ++j )
		{
		
			results[j].claimProbability += 
				expandedResults[ i*step + j ].claimProbability;
			results[j].defaultProbability += 
				expandedResults[ i*step + j ].defaultProbability;
			results[j].expectedClaim +=  
				expandedResults[ i*step + j ].expectedClaim;
			results[j].expectedLoss += 
				expandedResults[ i*step + j ].expectedLoss;
			results[j].expectedLossPercent += 
				expandedResults[ i*step + j ].expectedLossPercent;

			results[j].maxClaim = MAX( results[j].maxClaim, 
				expandedResults[ i*step + j ].maxClaim );
			results[j].maxClaimPv = MAX( results[j].maxClaimPv, 
				expandedResults[ i*step + j ].maxClaimPv );
		
		}
	
	}
	
	// correct
	if( correction == 1.0 )
	{
	
		return;
	
	}
	
	for( unsigned int j = 0; j < step; ++j )
	{
	
		results[j].claimProbability *= correction;
		results[j].defaultProbability *= correction;
		results[j].expectedClaim *= correction;
		results[j].expectedLoss *= correction;
		results[j].expectedLossPercent *= correction;

		results[j].maxClaim *= correction;
		results[j].maxClaimPv *= correction;
	
	}

}

void capModel3Cuda( CudaInvariants* invariants, CudaResult* results )
{

	// configuration
	hydrazine::cudaCheck( hipGetDevice( &invariants->device ) );

	// streams
	hipStream_t stream;
	hydrazine::cudaCheck( hipStreamCreate( &stream ) );
/*
	// symbols
	hydrazine::cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL( wtab), hostWtab, 128 * sizeof(FLOAT), 
		0, hipMemcpyHostToDevice ) );
	hydrazine::cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL( ytab), hostYtab, 128 * sizeof(FLOAT), 
		0, hipMemcpyHostToDevice ) );
	hydrazine::cudaCheck( hipMemcpyToSymbol(HIP_SYMBOL( ktab), hostKtab, 
		128 * sizeof(unsigned long), 
		0, hipMemcpyHostToDevice ) );	
*/
	hipDeviceProp_t properties;
	hydrazine::cudaCheck( hipGetDeviceProperties( &properties, 
		invariants->device ) );

	unsigned int totalMemory = properties.totalGlobalMem;
	unsigned int maxThreads = totalMemory - invariantSize( invariants, 0 );
	maxThreads = MIN( maxThreads, MAX_THREADS );
	maxThreads = MIN( maxThreads, 
		(unsigned int)properties.regsPerBlock / MIN_REGS );
	maxThreads = MIN( maxThreads, (unsigned int)properties.maxThreadsDim[0] );
	
	unsigned int maxCtas = MAX_CTAS;
	maxCtas = MIN( maxCtas, CEIL_DIV( invariants->parameters->chunkSize, 
		maxThreads ) );
	
	unsigned int threads = maxCtas * maxThreads;

	unsigned int chunkSize = invariants->parameters->chunkSize;

	invariants->parameters->chunkSize = 
		CEIL_DIV( invariants->parameters->chunkSize, threads );
	
	// memory
	CudaScratch* deviceScratch;
	CudaInvariants* hostInvariants;
	CudaInvariants* deviceInvariants;
	CudaResult* deviceResults;
	CudaResult* hostResults;
	
	unsigned int scratchBytes = threads * scratchSize( invariants );
	unsigned int invariantBytes = invariantSize( invariants, threads );
	unsigned int resultBytes = threads * 
		( invariants->counterPartySize + 1) * sizeof( CudaResult );
	
	hydrazine::cudaCheck( hipHostMalloc( (void**) &hostInvariants, invariantBytes ) );
	hydrazine::cudaCheck( hipHostMalloc( (void**) &hostResults, resultBytes ) );
	
	hydrazine::cudaCheck( hipMalloc( (void**) &deviceResults, resultBytes ) );
	hydrazine::cudaCheck( hipMalloc( (void**) &deviceScratch, scratchBytes ) );
	hydrazine::cudaCheck( hipMalloc( (void**) &deviceInvariants, invariantBytes ) );
	
	// compute iterations
	
	packInvariants( hostInvariants, invariants, threads );
	
	// copy
	hydrazine::cudaCheck( hipMemcpyAsync( deviceInvariants, hostInvariants, 
		invariantBytes, hipMemcpyHostToDevice, stream ) );
			
	// compute
	capModel3Kernel<<< maxCtas, maxThreads, 0, stream >>>( deviceInvariants, 
		deviceScratch, deviceResults );
	
	// copy
	hydrazine::cudaCheck( hipMemcpyAsync( hostResults, deviceResults, 
		resultBytes, hipMemcpyDeviceToHost, stream ) );
	
	FLOAT correctionFactor = ( chunkSize + 0.0 ) / 
		( threads * invariants->parameters->chunkSize );
	
	hydrazine::cudaCheck( hipStreamSynchronize( stream ) );
	
	// reduction
	reduceResults( results, hostResults, invariants, threads, 
		correctionFactor );
	
	// cleanup
	hydrazine::cudaCheck( hipStreamDestroy( stream ) );
	
	hydrazine::cudaCheck( hipHostFree( hostResults ) );
	hydrazine::cudaCheck( hipHostFree( hostInvariants ) );
	
	hydrazine::cudaCheck( hipFree( deviceInvariants ) );
	hydrazine::cudaCheck( hipFree( deviceScratch ) );
	hydrazine::cudaCheck( hipFree( deviceResults ) );

}

////////////////////////////////////////////////////////////////////////////////

#endif

