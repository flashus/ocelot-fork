

#include <hip/hip_runtime.h>
#include <iostream>

__noinline__ __host__ __device__ int fib(int n) {
    if (n <= 1) {
        return n;
    } else {
        return fib(n-1)+fib(n-2);
    }
}

__global__ void fibonacci( int* array )
{
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	array[id] = fib( array[id] );
}

#define N 64

int main()
{
	int* host = new int[N];
	int* reference = new int[N];
	int* device;
	hipMalloc( (void**)&device, N * sizeof( int ) );
	
	for( unsigned int i = 0; i < N; ++i )
	{
		host[i] = i;
		reference[i] = i;
	}
	
	hipMemcpy( device, host, N * sizeof( int ), hipMemcpyHostToDevice );

	fibonacci<<< 1, N >>>( device );

	hipMemcpy( host, device, N * sizeof( int ), hipMemcpyDeviceToHost );
	
	bool pass = true;
	
	for( unsigned int i = 0; i < N; ++i )
	{
		reference[i] = fib( reference[i] );
		if( reference[i] != host[i] )
		{
			std::cout << "At index " << i << " gpu version computed " 
				<< host[i] << " while cpu computed " << reference[i] << "\n";
			pass = false;
			break;
		}
	}

	if( pass )
	{
		std::cout << "TEST PASSED\n";
	}
	else
	{
		std::cout << "TEST FAILED\n";
	}

	hipFree( device );
	delete[] reference;	
	delete[] host;
}

