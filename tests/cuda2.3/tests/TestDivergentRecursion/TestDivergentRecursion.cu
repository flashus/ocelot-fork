
#include <hip/hip_runtime.h>
/*  \file TestDivergentRecursion.cu
	\author Gregory Diamos <gregory.diamos@gatech.edu>
	\date Tuesday November 9, 2010
	\brief A CUDA assembly test for short-circuiting control flow.
*/

const unsigned int threads = 512;

__device__ bool out[threads];

__device__ unsigned int divergent_function(unsigned int id)
{
/*	if(id == 1) return 0;
	
	unsigned int result = 0;
	
	if(id & 0x10)
	{
		result = divergent_function(id - 2);
	}
	else
	{
		result = divergent_function(id - 1);
	}
	
	return result;
*/

	goto FunctionEntryPoint;

	FunctionEntryPoint:
		if(id > 1)
		{
			if(id & 2)
			{
				id -= 2;		
				goto FunctionEntryPoint;			
			}
			else
			{
				id -= 1;
				goto FunctionEntryPoint;
			}
		}
		return max(1, id) - 1;
}

__global__ void divergent_recusion()
{
	out[threadIdx.x] = divergent_function(threadIdx.x);
}

int main(int argc, char** argv)
{
	divergent_recusion<<<1, threads>>>();
}


