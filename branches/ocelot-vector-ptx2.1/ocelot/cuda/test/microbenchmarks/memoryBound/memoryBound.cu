#include "hip/hip_runtime.h"
/*!
	\file memoryBound.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief test application measuring achieved device-to-device memory bandwidth 
*/

#define LINESIZE 64
#define BlockSize 4096

#include <hydrazine/implementation/Timer.h>
#include <stdio.h>
#include <assert.h>

#define cudaCheckCall(x) { hipError_t result = x; if (result != hipSuccess) \
	{ printf("Error: %s\n", hipGetErrorString(result)); assert(0); } }

#define min(a, b) ((a) > (b) ? (b) : (a))

/////////////////////////////////////////////////////////////////////////////////////////////////

extern "C" __global__ void kernel_Touch(
	float *buffer,
	float val,
	size_t elements) {
	
	int id = threadIdx.x + blockDim.x * blockIdx.x;
	for (; id < elements; id += blockDim.x) {
		buffer[id] = val;
	}
}

extern "C" __global__ void kernel_GlobalTransfer(
	float *destPtr,
	float *srcPtr,
	size_t threadBaseStride,
	size_t threadStride,
	size_t elementsPerThread) {
	
	int id = threadIdx.x * threadBaseStride + blockDim.x * blockIdx.x * elementsPerThread;
	destPtr += id;
	srcPtr += id;

	for (size_t i = 0; i < elementsPerThread; i++) {
		*destPtr = *srcPtr;
		destPtr += threadStride;
		srcPtr += threadStride;
	}	
}

/////////////////////////////////////////////////////////////////////////////////////////////////

void run(size_t elements, int ctaSize = 256) {
	float *A_dest, *A_src;
	size_t bytes = sizeof(float)*elements;
	int runs = 10;
	
	cudaCheckCall(hipMalloc((void **)&A_dest, bytes));
	cudaCheckCall(hipMalloc((void **)&A_src, bytes));
	
	//kernel_Touch<<< dim3(min(16, elements/64), 1), dim3(64, 1) >>>(A_src, 2.0f, elements);
	//kernel_Touch<<< dim3(min(16, elements/64), 1), dim3(64, 1) >>>(A_dest, 0.0f, elements);
	
	// dense copies (enables coalesced accesses)
	hydrazine::Timer timer;
	timer.start();
	for (int i = 0; i < runs; i++) {
		float *ptrs[2] = { A_dest, A_src };
		
		size_t baseStride = 1;
		size_t intrathreadStride = ctaSize;
		size_t elementsPerThread = 8*ctaSize;
		
		dim3 block(ctaSize, 1);
		dim3 grid(elements / (ctaSize) / elementsPerThread, 1);
		
		kernel_GlobalTransfer<<< grid, block >>>(ptrs[(i) % 2], ptrs[(i+1)%2], 
			baseStride, intrathreadStride, elementsPerThread);
	}
	hipDeviceSynchronize();
	timer.stop();
	double coalescedRuntime = timer.seconds();
	
	// dense copies (enables coalesced accesses)
	timer.start();
	for (int i = 0; i < runs; i++) {
		float *ptrs[2] = { A_dest, A_src };
		
		size_t elementsPerThread = 8*ctaSize;
		size_t baseStride = elementsPerThread;
		size_t intrathreadStride = 1;
		
		dim3 block(ctaSize, 1);
		dim3 grid(elements / (ctaSize) / elementsPerThread, 1);
		
		kernel_GlobalTransfer<<< grid, block >>>(ptrs[(i) % 2], ptrs[(i+1)%2], 
			baseStride, intrathreadStride, elementsPerThread);
	}
	hipDeviceSynchronize();
	timer.stop();
	double sequentialRuntime = timer.seconds();
	
	hipFree(A_dest);
	hipFree(A_src);
	
	double coalescedBandwidth = (double)(elements >> 18) * 2 * runs / (coalescedRuntime * 1000.0) ;
	double serialBandwidth = (double)(elements >> 18) * 2 * runs / (sequentialRuntime * 1000.0);
	
	printf("allocation size: %d MB. Coalesced bandwidth: %f GB/s. Sequential bandwidth: %f GB/s\n",
		(elements>>18), coalescedBandwidth, serialBandwidth);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char *arg[]) {

	run((100 << 20));
	
	return 0;
}

