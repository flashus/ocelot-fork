/*!
	\file TestConvergence.cu
	\date 17 February 2011
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief Demonstrates convergent execution
*/

// Standard Library Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

extern "C" __global__ void convergence(float *A) {
	int i = threadIdx.x;
	A[i] = i * 2.0f + 1.0f;;
}

static void testConvergence() {
	const int N = 4;
	float *A_host, *A_device;
	size_t bytes = N * sizeof(float);
	
	A_host = (float *)malloc(bytes);
	hipMalloc((void **)&A_device, bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = (float)(i+1);
	}
	hipMemcpy(A_device, A_host, bytes, hipMemcpyHostToDevice);
	
	convergence<<< dim3(1,1), dim3(N, 1, 1) >>>(A_device);
	
	hipMemcpy(A_host, A_device, bytes, hipMemcpyDeviceToHost);
	
	int errors = 0;
	for (int i = 0; (errors < 5) && i < N; i++) {
		float expected = /* (float)(i+1) + */ 2.0f * i + 1.0f;
		float got = A_host[i];
		if (fabs(expected - got) > 0.001f) {
			++errors;
			printf("error [%d] - expected: %f, got %f\n", i, expected, got);
		}
	}
	
	printf("Test %s\n", (errors ? "FAILED": "Passed"));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv )
{
	testConvergence();

	return 0;
}
