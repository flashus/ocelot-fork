/*!
	\file TestConvergence.cu
	\date 17 February 2011
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief Demonstrates divergent execution
*/

// Standard Library Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define INCLUDE_TRANSPOSE 1

/////////////////////////////////////////////////////////////////////////////////////////////////

extern "C" __global__ void divergence(float *A) {
	int i = threadIdx.x;
	if (i & 0x01) {
		A[i] = i * 2.0f + 1.0f;
	}
	else {
		A[i] *= 1.25f;
	}
}

#if INCLUDE_TRANSPOSE
/*!
	\brief computes the out-of-place transpose of an M-by-N matrix where M and N are divible by 4
*/
extern "C" __global__ void transpose(float *A, const float *B, int M, int N) {
	__shared__ float block[4][5];
	
	//
	// copy the block B[ by:by+4, bx:bx+4 ] to A[ bx:bx+4, by:by+4 ]
	//
	
	int bi = blockIdx.x * 4 + threadIdx.x + blockIdx.y * 4 * N;
	
	for (int i = 0; i < 4; i++) {
		block[i][threadIdx.x] = B[bi + i * N];
	}
	__syncthreads();
	
	//
	// A is N-by-M
	//
	int ai = blockIdx.y * 4 + threadIdx.x + blockIdx.x * 4 * M;
	for (int i = 0; i < 4; i++) {		
		A[ai + i * M] = block[threadIdx.x][i];
	}
}
#endif
/////////////////////////////////////////////////////////////////////////////////////////////////

static float expectedValue(int n) {
	if (n & 0x01) {
		return n * 2.0f + 1.0f;
	}
	else {
		return 1.25f * ((float)n + 1.0f);
	}
}

static int testDivergence() {
	const int N = 4;
	float *A_host, *A_device;
	size_t bytes = N * sizeof(float);
	
	A_host = (float *)malloc(bytes);
	hipMalloc((void **)&A_device, bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = (float)(i+1);
	}
	hipMemcpy(A_device, A_host, bytes, hipMemcpyHostToDevice);
	
	divergence<<< dim3(1,1), dim3(N, 1, 1) >>>(A_device);
	
	hipMemcpy(A_host, A_device, bytes, hipMemcpyDeviceToHost);
	
	int errors = 0;
	for (int i = 0; (errors < 5) && i < N; i++) {
		float expected = expectedValue(i);
		float got = A_host[i];
		if (fabs(expected - got) > 0.001f) {
			++errors;
			printf("error [%d] - expected: %f, got %f\n", i, expected, got);
		}
	}
	
	hipFree(A_device);
	free(A_host);
	
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

#if INCLUDE_TRANSPOSE
static int testTranspose() {
	const int M = 4;
	const int N = 4;
	
	float *A_gpu, *A_gpu2, *A_host;
	size_t bytes = sizeof(float)*M*N;
	hipMalloc((void **)&A_gpu, bytes);
	hipMalloc((void **)&A_gpu2, bytes);
	A_host = (float *)malloc(bytes);
	
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			A_host[i * N + j] = (float)(i * N) + (float)j;
		}
	}
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);
	
	dim3 grid(M / 4, N / 4);
	dim3 block(4, 1);
	transpose<<< grid, block >>>(A_gpu2, A_gpu, M, N);
	hipMemcpy(A_host, A_gpu2, bytes, hipMemcpyDeviceToHost);
	
	// A is N-by-M
	int errors = 0;
	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++) {
			float expected = (float)(j * N) + (float)i;
			float got = A_host[i * M + j];
			if (fabs(expected - got) > 0.001f) {
				++errors;
				printf("error [%d, %d] - expected: %f, got: %f\n", i, j, expected, got);
			}
			if (errors >= 5) {
				goto testTranspose_exit;
			}
		}
	}
testTranspose_exit:
	hipFree(A_gpu);
	hipFree(A_gpu2);
	free(A_host);
	
	return errors;
}
#endif

/////////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv )
{
	int errors = 0;
	if (!errors) {
		errors = testDivergence();
	}
#if INCLUDE_TRANSPOSE
	if (!errors) {
		errors = testTranspose();
	}
#endif

	printf("Test %s\n", (errors ? "FAILED": "Passed"));
	return 0;
}
