/*!
	\file TestConvergenceLoop.cu
	\date 17 February 2011
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief Demonstrates convergent execution
*/

// Standard Library Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

/*!

*/
extern "C" __global__ void convergenceWithLoop(float *A, int P) {
	int i = threadIdx.x;
	float f = i * 2.0f + 1.0f;;
	for (int p = 0; p < P; p++) {
		f *= 1.25f - 2.0f;
	}
	A[i] = f;
}

/*!
	\brief loop with divergent control flow
*/
extern "C" __global__ void loopEarlyExit(float *A, float dt) {
	int i = threadIdx.x;
	float f = A[i];
	for (int p = 0; p < i + 1; p++) {
		f += dt;
	}
	A[i] = f;
}

/*!
	\brief sums the elements of an array
*/
extern "C" __global__ void reduction(float *A, int N) {
	__shared__ float buffer[8];
	
	int idx = threadIdx.x;
	float sum = 0.0;
	
	for (int i = 0; i < N; i+= blockDim.x, idx += blockDim.x) {
		buffer[threadIdx.x] = 0;
		if (idx < N) {
			buffer[threadIdx.x] = A[idx];
		}
		__syncthreads();
		for (int j = 0; j < 8; j++) {
			sum += buffer[j];
		}
		__syncthreads();
	}
	A[threadIdx.x] = sum;
}

static float expectedValue(int n,  int P) {
	int i = n;
	float f = i * 2.0f + 1.0f;;
	for (int p = 0; p < P; p++) {
		f *= 1.25f - 2.0f;
	}
	return f;
}

static int testConvergenceWithLoop() {
	const int N = 4;
	const int P = 3;
	float *A_host, *A_device;
	size_t bytes = N * sizeof(float);
	
	A_host = (float *)malloc(bytes);
	hipMalloc((void **)&A_device, bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = (float)(i+1);
	}
	hipMemcpy(A_device, A_host, bytes, hipMemcpyHostToDevice);
	
	convergenceWithLoop<<< dim3(1,1), dim3(N, 1, 1) >>>(A_device, P);
	
	hipMemcpy(A_host, A_device, bytes, hipMemcpyDeviceToHost);
	
	int errors = 0;
	for (int i = 0; (errors < 5) && i < N; i++) {
		float expected = expectedValue(i, P);
		float got = A_host[i];
		if (fabs(expected - got) > 0.001f) {
			++errors;
			printf("error 1 [%d] - expected: %f, got %f\n", i, expected, got);
		}
	}
	
	hipFree(A_device);
	free(A_host);
	return errors;
}

static int testLoopEarlyExit() {
	const int N = 8;
	float *A_host, *A_gpu;
	
	size_t bytes = sizeof(float)*N;
	hipMalloc((void **)&A_gpu, bytes);
	A_host = (float *)malloc(bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = 2.0f * (float)i / (float)(N-1);
	}
	
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);
	
	float dt = 0.25f;
	dim3 grid(1,1);
	dim3 block(N,1,1);
	
	loopEarlyExit<<< grid, block >>>(A_gpu, dt);

	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	int errors = 0;
	for (int i = 0; errors < 5 && i < N; i++) {
		float got = A_host[i];
		float expected = 2.0f * (float)i / (float)(N-1) + (i+1) * dt;
		if (fabs(got - expected) > 0.0001f) {
			printf("ERROR 2 [%d] - expected: %f, got: %f\n", i, expected, got);
			++errors;
		}
	}
	
	free(A_host);
	hipFree(A_gpu);
	
	return errors;
}

static int testReduction() {
	const int N = 32;
	float *A_host, *A_gpu;
	
	size_t bytes = sizeof(float)*N;
	hipMalloc((void **)&A_gpu, bytes);
	A_host = (float *)malloc(bytes);
	
	float sum = 0.0f;
	for (int i = 0; i < N; i++) {
		A_host[i] = 2.0f * (float)i / (float)(N-1);
		sum += A_host[i];
	}
	
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);
	
	dim3 grid(1,1);
	dim3 block(8,1,1);
	
	reduction<<< grid, block >>>(A_gpu, N);

	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	int errors = 0;
	
	if (fabs(A_host[0] - sum) > 0.001f) {
		++errors;
		printf("ERROR 3 - expected sum: %f, got: %f\n", sum, A_host[0]);
	}
	
	free(A_host);
	hipFree(A_gpu);
	
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv )
{
	int errors = 0;
	/*
	if (!errors) {
		errors += testConvergenceWithLoop();
	}
	if (!errors) {
		errors += testLoopEarlyExit();
	}
	*/
	if (!errors) {
		errors += testReduction();
	}

	printf("Pass/Fail : %s\n", (errors ? "Fail":"Pass"));

	return 0;
}
