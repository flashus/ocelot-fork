/*!
	\file TestConvergenceLoop.cu
	\date 17 February 2011
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief Demonstrates convergent execution
*/

// Standard Library Includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

extern "C" __global__ void convergenceWithLoop(float *A, int P) {
	int i = threadIdx.x;
	float f = i * 2.0f + 1.0f;;
	for (int p = 0; p < P; p++) {
		f *= 1.25f - 2.0f;
	}
	A[i] = f;
}

static float expectedValue(int n,  int P) {
	int i = n;
	float f = i * 2.0f + 1.0f;;
	for (int p = 0; p < P; p++) {
		f *= 1.25f - 2.0f;
	}
	return f;
}

static void testConvergenceWithLoop() {
	const int N = 4;
	const int P = 3;
	float *A_host, *A_device;
	size_t bytes = N * sizeof(float);
	
	A_host = (float *)malloc(bytes);
	hipMalloc((void **)&A_device, bytes);
	
	for (int i = 0; i < N; i++) {
		A_host[i] = (float)(i+1);
	}
	hipMemcpy(A_device, A_host, bytes, hipMemcpyHostToDevice);
	
	convergenceWithLoop<<< dim3(1,1), dim3(N, 1, 1) >>>(A_device, P);
	
	hipMemcpy(A_host, A_device, bytes, hipMemcpyDeviceToHost);
	
	int errors = 0;
	for (int i = 0; (errors < 5) && i < N; i++) {
		float expected = expectedValue(i, P);
		float got = A_host[i];
		if (fabs(expected - got) > 0.001f) {
			++errors;
			printf("error [%d] - expected: %f, got %f\n", i, expected, got);
		}
	}
	
	printf("Pass/Fail : %s\n", (errors ? "Fail":"Pass"));
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main( int argc, char** argv )
{
	testConvergenceWithLoop();

	return 0;
}
