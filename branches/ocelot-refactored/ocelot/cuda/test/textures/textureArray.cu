#include "hip/hip_runtime.h"
/*!
	\file textureArray.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief tests implementation of hipMallocArray() and hipBindTextureToArray()
	\date February 12, 2010

	This was taken directly from the NVIDIA CUDA Programming Guide
*/

//////////////////////////////////////////////////////////////////////////////////////////////////

#include <iostream>

//////////////////////////////////////////////////////////////////////////////////////////////////

// 2D float texture
texture<float, 2, hipReadModeElementType> texRef;

// Simple transformation kernel
__global__ void transformKernel(float* output, int width, int height, float theta) {
  // Calculate normalized texture coordinates
  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  float u = x / (float)width;
  float v = y / (float)height;
  // Transform coordinates
  u -= 0.5f;
  v -= 0.5f;
  float tu = u * cosf(theta) – v * sinf(theta) + 0.5f;
  float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;
  // Read from texture and write to global memory
  output[y * width + x] = tex2D(tex, tu, tv);
}

//////////////////////////////////////////////////////////////////////////////////////////////////

// Host code
int main(int argc, char *arg[]) {

	// Allocate CUDA array in device memory
	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32,0,0,0, hipChannelFormatKindFloat);
	hipArray* cuArray;
	hipMallocArray(&cuArray, &channelDesc, width, height);

	// Copy to device memory some data located at address h_data
	// in host memory
	hipMemcpyToArray(cuArray, 0, 0, h_data, size, hipMemcpyHostToDevice);

	// Set texture parameters
	texRef.addressMode[0] = hipAddressModeWrap;
	texRef.addressMode[1] = hipAddressModeWrap;
	texRef.filterMode     = hipFilterModeLinear;
	texRef.normalized     = true;

	// Bind the array to the texture
	hipBindTextureToArray(texRef, cuArray, channelDesc);

	// Allocate result of transformation in device memory
	float* output;
	hipMalloc((void**)&output, width * height * sizeof(float));

	// Invoke kernel
	dim3 dimBlock(16, 16);
	dim3 dimGrid((width + dimBlock.x – 1) / dimBlock.x,
		           (height + dimBlock.y – 1) / dimBlock.y);
	transformKernel<<<dimGrid, dimBlock>>>(output, width, height,  angle);

	// Free device memory
	hipFreeArray(cuArray);
	hipFree(output);

	return 0;
}

