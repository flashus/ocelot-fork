#include "hip/hip_runtime.h"
/*!
	\brief sequence.cu
	\author Andrew Kerr

	\brief simple test of a CUDA implementation's ability to allocate memory on the device, launch
		a kernel, and fetch its results
*/

#include <stdio.h>

extern "C" __global__ void sequence(int *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < N) {
		A[i] = 2*i;
	}
}


int main(int argc, char *arg[]) {

	const int N = 1024;
	int *A_host, *A_gpu;
	int errors = 0;

	size_t bytes = sizeof(int)*N;

	if (hipMalloc((void **)A_gpu, bytes) != hipSuccess) {
		printf("hipMalloc() - failed to allocate %d bytes on device\n", bytes);
		return -1;
	}

	A_host = (int *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		A_host[i] = -1;
	}
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);

	sequence<<< grid, block >>>(A_gpu, N);

	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		if (A_host[i] != 2*i) {
			++errors;
		}
	}

	hipFree(A_gpu);
	free(A_host);

	if (errors) {
		printf("FAILED\n");
	}
	else {
		printf("PASSED\n");
	}

	return 0;
}

