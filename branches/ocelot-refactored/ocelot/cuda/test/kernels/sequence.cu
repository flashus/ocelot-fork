#include "hip/hip_runtime.h"
/*!
	\brief sequence.cu
	\author Andrew Kerr

	\brief simple test of a CUDA implementation's ability to allocate memory on the device, launch
		a kernel, and fetch its results
*/

#include <stdio.h>

extern "C" __global__ void sequence(int *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < N) {
		A[i] = 2*i;
	}
}

extern "C" __global__ void test_shr(int *A, const int *B) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int b;
	__shared__ int storage[256];
	
	storage[threadIdx.x] = B[i];
	__syncthreads();
	if (i & 1) {
		b = storage[threadIdx.x ^ 1] * 2;
	}
	else {
		b = storage[threadIdx.x ^ 1] * 3;
	}
	A[i] = b;
}

int main(int argc, char *arg[]) {

	const int N = 1024;
	int *A_host, *A_gpu;
	int errors = 0;

	size_t bytes = sizeof(int)*N;

	if (hipMalloc((void **)A_gpu, bytes) != hipSuccess) {
		printf("hipMalloc() - failed to allocate %d bytes on device\n", bytes);
		return -1;
	}

	A_host = (int *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		A_host[i] = -1;
	}
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);

	sequence<<< grid, block >>>(A_gpu, N);

	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);

	for (int i = 0; i < N; i++) {
		if (A_host[i] != 2*i) {
			++errors;
		}
	}
	
	int *B_gpu = 0;
	if (hipMalloc((void **)&B_gpu, bytes) != hipSuccess) {
		printf("hipMalloc() - failed to allocate %d bytes on device\n", bytes);
		hipFree(A_gpu);
		free(A_host);
		return -1;
	}
	
	sequence<<< grid, block >>>(A_gpu, N);
	testShr<<< grid, block >>>(B_gpu, A_gpu);
	
	if (hipMemcpy(A_host, B_gpu, bytes, hipMemcpyDeviceToHost) != hipSuccess) {
		printf("hipMemcpy(A, B) - failed to copy %d bytes from device to host\n",	
			bytes);
		hipFree(A_gpu);
		hipFree(B_gpu);
		free(A_host);
	}
	
	for (int i = 0; (errors < 5) && i < N; ++) {
		int b;
		if (i & 1) {
			b = (i ^ 1) * 2 * 2;
		}
		else {
			b = (i ^ 1) * 2 * 3;
		}
		int got = A_host[i];
		if (b != got) {
			printf("ERROR 1 [%d] - expected: %d, got: %d\n", i, b, got);
			++errors;
		}
	}

	hipFree(B_gpu);
	hipFree(A_gpu);
	free(A_host);

	if (errors) {
		printf("FAILED\n");
	}
	else {
		printf("PASSED\n");
	}

	return 0;
}

