/*!
	\file TestMemoryTraceGenerator_kernel.cu
	
	\author Andrew Kerr <arkerr@gatech.edu>
	
	\brief defines kernels used in the TestMemoryTraceGenerator application
	
	You may run this kernel in a standalone fashion as follows:
	
		nvcc --run TestMemoryTraceGenerator_kernel.cu -DSTANDALONE
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

//#include <cuda_runtime.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

#define k_mv_SIZE 64
__global__ void k_mv_product(int M, int N, const float *A_gpu, const float *V_gpu, float *R_gpu) {
	unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ float V_shared[k_mv_SIZE];
	float r_sum = 0;
	
	V_gpu += threadIdx.x;
	A_gpu += row;
	R_gpu += row;
	
	for (int j = 0; j < N; j += k_mv_SIZE) {
		V_shared[threadIdx.x] = *V_gpu;
		__syncthreads();
		#pragma unroll
		for (int k = 0; k < k_mv_SIZE; k++) {
			r_sum += *A_gpu * V_shared[k];
			A_gpu += M;
		}
		V_gpu += k_mv_SIZE;
		__syncthreads();
	}
	
	*R_gpu = r_sum;
}

void matrix_vector_product(int M, int N, const float *A_gpu, const float *V_gpu, float *R_gpu) {
	if (!(N % k_mv_SIZE) && !(M % k_mv_SIZE)) {
		dim3 grid(M / k_mv_SIZE, 1);
		dim3 block(k_mv_SIZE, 1);
		k_mv_product<<< grid, block >>>(M, N, A_gpu, V_gpu, R_gpu);
	}
	else {
		printf("matrix_vector_product(%d, %d, ..) - matrix dimensions must be multiples of %d\n", M, N, k_mv_SIZE);
	}
}

/////////////////////////////////////////////////////////////////////////////////////////////////

#define signum(x) ((x) < 0 ? -1 : 1)

extern "C" int test_entry() {
	const int M = 256, N = 128;
	
	int i, j;
	int result = 0;
	
	float *A_gpu, *A_cpu, *V_gpu, *V_cpu, *R_gpu, *R_cpu;
	float sq_error;
	
	size_t A_size = M * N * sizeof(float);
	size_t V_size = N * sizeof(float);
	size_t R_size = M * sizeof(float);
	
	// allocate memory
	if (hipMalloc((void **)&A_gpu, A_size) != hipSuccess) {
		printf("failed to allocate %u bytes for A_gpu\n", (unsigned int)A_size);
		return -1;
	}
	if (hipMalloc((void **)&V_gpu, V_size) != hipSuccess) {
		printf("failed to allocate %u bytes for V_gpu\n", (unsigned int)V_size);
		return -1;
	}
	if (hipMalloc((void **)&R_gpu, R_size) != hipSuccess) {
		printf("failed to allocate %u bytes for R_gpu\n", (unsigned int)R_size);
		return -1;
	}
	
	hipHostMalloc((void **)&A_cpu, A_size, hipHostMallocDefault);
	hipHostMalloc((void **)&V_cpu, V_size, hipHostMallocDefault);
	hipHostMalloc((void **)&R_cpu, R_size, hipHostMallocDefault);
	
	// initialize data
	for (j = 0; j < N; j++) {
		for (i = 0; i < M; i++) {
			A_cpu[i + j * M] = 1.0f / (float)(1 + abs(i - j)) * (float)signum(i - j);
			R_cpu[i] = -1;
		}
		V_cpu[j] = 1.0f + (float)(j % 4);
	}
	hipMemcpy(A_gpu, A_cpu, A_size, hipMemcpyHostToDevice);
	hipMemcpy(V_gpu, V_cpu, V_size, hipMemcpyHostToDevice);
	hipMemcpy(R_gpu, R_cpu, R_size, hipMemcpyHostToDevice);
	
	// invoke kernel
	matrix_vector_product(M, N, A_gpu, V_gpu, R_gpu);
	
	// validate result
	hipMemcpy(R_cpu, R_gpu, R_size, hipMemcpyDeviceToHost);
	
	sq_error = 0;
	for (i = 0; i < M; i++) {
		float sum = 0;
		for (j = 0; j < N; j++) {
			sum += A_cpu[i + j * M] * V_cpu[j];
		}
		sum = R_cpu[i] - sum;
		sq_error += sum * sum;
	}
	if (sq_error > 0.01) {
		result = -1;
	}
	
	// cleanup
	hipHostFree(A_cpu);
	hipHostFree(V_cpu);
	hipHostFree(R_cpu);
	
	hipFree(A_gpu);
	hipFree(V_gpu);
	hipFree(R_gpu);
	
	return result;
}

#ifdef STANDALONE
int main() {
	int result = test_entry();
	if (result == 0) {
		printf("test succeeded\n");
	}
	else {
		printf("TEST FAILED\n");
	}
	return 0;
}
#endif

