#include "hip/hip_runtime.h"
/*!
	\file ComputeBound.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\date Nov 3, 2010
*/

#include <sys/time.h>
#include <stdio.h>

#define Iterations 1024

#define BLOCK1 { p1 = p1 * a + b; p2 = p2 * b + c; p3 = p3 * c + d; p4 = p4 * d + e; }
#define BLOCK2 { p1 = p1 * a + b; q1 = q1 + a; p2 = p2 * b + c; q2 = q2 + b; \
	p3 = p3 * c + d; q3 = q3 + c; p4 = p4 * d + e; q4 = q4 + d; }
#define BLOCK3 { p1 = p1 * a; p2 = p2 * b; p3 = p3 * c; p4 = p4 * d; }
#define BLOCK4 { p1 = p1 * a + a; p2 = p2 * b + b; p3 = p3 * c + c; p4 = p4 * d + d; }

#define BLOCKN BLOCK1

#define GROUP1 BLOCKN BLOCKN BLOCKN BLOCKN
#define GROUP2 GROUP1 GROUP1 GROUP1 GROUP1
#define GROUP4 GROUP2 GROUP2 GROUP2 GROUP2
#define GROUP8 GROUP4 GROUP4 GROUP4 GROUP4

#define MADs 2048
#define ADDs 0
#define MULs 0
extern "C" __global__ void FloatComputeBound(
	float *A, 
	const float a, const float b, const float c, const float d, const float e, bool store) {
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	
	float q1 = A[tid] - 1;
	float q2 = A[tid] * 0.75f;
	float q3 = A[tid] * 0.25f;
	float q4 = A[tid] * 0.5f;
	
	float p1 = A[tid] - 1;
	float p2 = A[tid] * 1.25f;
	float p3 = A[tid] * 0.75f;
	float p4 = A[tid] * 0.5f;
	
	for (int i = 0; i < Iterations; i++) {
		GROUP8
		GROUP8
	}
	
	if (store) {
		A[tid] = p1 - q1 + p2 - q2 + p3 - q3 + p4 - q4;
	}
}

void run(const int M) {
	
	float *A_gpu, *A_cpu;
	const int N = 576;
	
	const int K = M * N;
	size_t bytes = sizeof(float)*K;
	
	hipMalloc((void **)&A_gpu, bytes);
	A_cpu = (float *)malloc(bytes);
	for (int i = 0; i < K; i++) {
		A_cpu[i] = (float)i / (float)(K - 1);
	}
	hipMemcpy(A_gpu, A_cpu, bytes, hipMemcpyHostToDevice);
	
	struct timeval startTime, endTime;
	
	hipDeviceSynchronize();
	gettimeofday(&startTime, 0);
	FloatComputeBound<<< dim3(M,1), dim3(N, 1) >>>(A_gpu, 1.125f, 0.125f, 0.5f, 1.25f, 1.25f, false);
	hipDeviceSynchronize();
	gettimeofday(&endTime, 0);
	
	double ms = (((double)endTime.tv_sec - (double)startTime.tv_sec) * 1000.0 + 
		((double)endTime.tv_usec - (double)startTime.tv_usec) / 1000.0);
	
	double s = ms / 1000.0;
	double GFLOPs = (2 * MADs + ADDs + MULs) / 1.0e6 * Iterations * K / 1.0e3;
	double GFLOPsPerSec = (GFLOPs / s);
	
	// CTAs, GFLOPs/second, GFLOPs, ms
	printf("%d, %f\n", M, GFLOPsPerSec );
	
	free(A_cpu);
	hipFree(A_gpu);
}

int main() {
	for (int i = 1; i < 50; i++) {
		run(i);
	}
	return 0;
}

