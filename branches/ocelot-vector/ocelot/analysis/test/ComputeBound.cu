#include "hip/hip_runtime.h"
/*!

*/

#include <sys/time.h>
#include <stdio.h>

#define Iterations 1024

// #define EMITCPUKERNEL

#ifdef EMITCPUKERNEL
#define MADs 256
#define ADDs 132
#define MULs 0
extern "C" __global__ void FloatComputeBound(
	float *A, 
	const float a, const float b, const float c, const float d, const float e, bool store) {
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;	
	const float f = (float)tid;
	
	float t = A[tid] - 9;
	float u = A[tid] * 0.25f;
	float v = A[tid] - 1.25f;
	float w = A[tid];
	
	float q = A[tid] * 0.75f;
	float x = A[tid] * 1.25f;
	float y = A[tid] * 0.5f;
	float z = A[tid];
	
	for (int i = 0; i < Iterations; i++) {
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	x = x * b + c; u = u + d;
	y = y * c + d; 
	x = x * c + d; u = u + e;
	y = y * d + e; 
	
	}
	
	if (store) {
	// out
	A[tid] = q - t + x - u + y - v + w - z;
	}
}

#else	

// ifndef EMITCPUKERNEL
#define MADs 320
#define ADDs 328
#define MULs 0
extern "C" __global__ void FloatComputeBound(
	float *A, 
	const float a, const float b, const float c, const float d, const float e, bool store) {
	
	int tid = threadIdx.x + blockDim.x * blockIdx.x;	
	const float f = (float)tid;
	
	float t = A[tid] - 9;
	float u = A[tid] * 0.25f;
	float v = A[tid] - 1.25f;
	float w = A[tid];
	
	float q = A[tid] * 0.75f;
	float x = A[tid] * 1.25f;
	float y = A[tid] * 0.5f;
	float z = A[tid];
	
	for (int i = 0; i < Iterations; i++) {


	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;

	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;

	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;

	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	
	q = q * a + b; t = t + c;
	x = x * b + c; u = u + d;
	y = y * c + d; v = v + e;
	z = z * d + e; w = w + a;	
	q = q * b + c; t = t + d;
	x = x * c + d; u = u + e;
	y = y * d + e; v = v + f;
	z = z * e + f; w = w + a;
	}
	
	if (store) {
		// out
		A[tid] = q - t + x - u + y - v + w - z;
	}
}
#endif

void run(const int M) {
	
	float *A_gpu, *A_cpu;
	const int N = 576;
	
	const int K = M * N;
	size_t bytes = sizeof(float)*K;
	
	hipMalloc((void **)&A_gpu, bytes);
	A_cpu = (float *)malloc(bytes);
	for (int i = 0; i < K; i++) {
		A_cpu[i] = (float)i / (float)(K - 1);
	}
	hipMemcpy(A_gpu, A_cpu, bytes, hipMemcpyHostToDevice);
	
	struct timeval startTime, endTime;
	
	// const float d, const float c, const float f, const float g, const float h
	hipDeviceSynchronize();
	gettimeofday(&startTime, 0);
	FloatComputeBound<<< dim3(M,1), dim3(N, 1) >>>(A_gpu, 1.125f, 0.125f, 0.5f, 1.25f, 1.25f, false);
	hipDeviceSynchronize();
	gettimeofday(&endTime, 0);
	
	double ms = (((double)endTime.tv_sec - (double)startTime.tv_sec) * 1000.0 + 
		((double)endTime.tv_usec - (double)startTime.tv_usec) / 1000.0);
	
	double s = ms / 1000.0;
	double GFLOPs = (2 * MADs + ADDs + MULs) / 1.0e6 * Iterations * K / 1.0e3;
	
	double GFLOPsPerSec = (GFLOPs / s);
	
	/*
	printf("M = %d\n", M);
	printf("Runtime: %f ms\n", ms);
	printf("GFLOPs: %f GFLOP/s per CTA\n\n", GFLOPs);
	*/
	printf("%d, %f, %f, %f\n", M, GFLOPsPerSec, GFLOPs, ms );
	
	free(A_cpu);
	hipFree(A_gpu);
}

int main() {
	for (int i = 1; i < 50; i++) {
		run(i);
	}
	return 0;
}

