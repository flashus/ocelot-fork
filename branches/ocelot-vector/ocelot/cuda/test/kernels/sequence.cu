/*!
	\brief sequence.cu
	\author Andrew Kerr

	\brief simple test of a CUDA implementation's ability to allocate memory on the device, launch
		a kernel, and fetch its results. One kernel requires no syncthreads, another kernel requires
		one synchronization
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void sequence(int *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i < N) {
		A[i] = 2*i;
	}
}

extern "C" __global__ void testShr(int *A, const int *B) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int b;
	__shared__ int storage[256];
	
	storage[threadIdx.x] = B[i];
	__syncthreads();
	if (i & 1) {
		b = storage[threadIdx.x ^ 1] * 2;
	}
	else {
		b = storage[threadIdx.x ^ 1] * 3;
	}
	A[i] = b;
}

int main(int argc, char *arg[]) {

	const int N = 1024;
	int *A_host, *A_gpu =0;
	int errors = 0;

	size_t bytes = sizeof(int)*N;

	if (hipMalloc((void **)&A_gpu, bytes) != hipSuccess) {
		printf("cudaMalloc() - failed to allocate %d bytes on device\n", (int)bytes);
		return -1;
	}

	A_host = (int *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		A_host[i] = -1;
	}
	
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);

	dim3 grid((N+31)/32,1);
	dim3 block(32, 1);
	sequence<<< grid, block >>>(A_gpu, N);
	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);
	for (int i = 0; i < N; i++) {
		if (A_host[i] != 2*i) {
			++errors;
		}
	}

	int *B_gpu = 0;
	if (hipMalloc((void **)&B_gpu, bytes) != hipSuccess) {
		printf("cudaMalloc() - failed to allocate %d bytes on device\n", (int)bytes);
		hipFree(A_gpu);
		free(A_host);
		return -1;
	}
	
	sequence<<< grid, block >>>(A_gpu, N);
	testShr<<< grid, block >>>(B_gpu, A_gpu);
	
	if (hipMemcpy(A_host, B_gpu, bytes, hipMemcpyDeviceToHost) != hipSuccess) {
		printf("cudaMemcpy(A, B) - failed to copy %d bytes from device to host\n", (int)bytes);
		hipFree(A_gpu);
		hipFree(B_gpu);
		free(A_host);
	}
	
	for (int i = 0; (errors < 5) && i < N; ++i) {
		int b;
		if (i & 1) {
			b = (i ^ 1) * 2 * 2;
		}
		else {
			b = (i ^ 1) * 2 * 3;
		}
		int got = A_host[i];
		if (b != got) {
			printf("ERROR 1 [%d] - expected: %d, got: %d\n", i, b, got);
			++errors;
		}
	}

	hipFree(B_gpu);
	hipFree(A_gpu);
	free(A_host);

	if (errors) {
		printf("FAILED\n");
	}
	else {
		printf("PASSED\n");
	}

	return 0;
}

