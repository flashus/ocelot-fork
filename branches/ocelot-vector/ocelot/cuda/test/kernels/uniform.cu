/*!

*/


#include <hip/hip_runtime.h>
#include <stdio.h>

/*!
	\brief simple kernel with uniform control flow
*/
extern "C" __global__ void kernel_uniform(float *A, float a) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	A[tid] = a * tid;
}

/*!
	\brief simple kernel with uniform control flow and a loop
*/
extern "C" __global__ void kernel_uniform_loop(float *A, float a, float *B, int b) {
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	A[tid] = a * tid;
	for (int i = 0; i < b; i++) {
		A[tid] += B[i];
	}
}

/*!

*/
int main() {
	const int N = 64;
	dim3 blockSize(32, 1);
	dim3 gridSize(N / blockSize.x, 1);

	float *A_gpu, *A_cpu, *B_cpu, *B_gpu;
	const int B = 5;

	size_t A_bytes = sizeof(float) * N;
	size_t B_bytes = sizeof(float) * B;

	A_cpu = (float *)malloc(A_bytes);
	B_cpu = (float *)malloc(B_bytes);
	hipMalloc((void **)&A_gpu, A_bytes);
	hipMalloc((void **)&B_gpu, B_bytes);

	for (int i = 0; i < N; i++) {
		A_cpu[i] = 0;
		if (i < B) {
			B_cpu[i] = 3 * i;
		}
	}

	hipMemcpy(A_gpu, A_cpu, A_bytes, hipMemcpyHostToDevice);
	hipMemcpy(B_gpu, B_cpu, B_bytes, hipMemcpyHostToDevice);

	int errors = 0;
	float A_k = 3;

	if (!errors) {
		kernel_uniform<<< gridSize, blockSize >>>(A_gpu, A_k);
		hipMemcpy(A_cpu, A_gpu, A_bytes, hipMemcpyDeviceToHost);

		for (int i = 0; i < N; i++) {
			float expected = A_k * i;
			float got = A_cpu[i];
			if (fabs(expected - got) > 0.00001f) {
				++errors;
				printf("ERROR in kernel_uniform() - [%d] - expected: %f, got: %f\n", i, expected, got);
				if (errors >= 5) goto cleanup;
			}
		}
	}

	if (!errors) {
		kernel_uniform_loop<<< gridSize, blockSize >>>(A_gpu, A_k, B_gpu, B);
		hipMemcpy(A_cpu, A_gpu, A_bytes, hipMemcpyDeviceToHost);

		for (int i = 0; i < N; i++) {
			int expected = A_k * i;
			for (int b = 0; b < B; b++) {
				expected += B_cpu[b];
			}
			int got = A_cpu[i];
			if (expected != got) {
				++errors;
				printf("ERROR in kernel_uniform_loop() - [%d] - expected: %d, got: %d\n", i, expected, got);
				if (errors >= 5) goto cleanup;
			}
		}
	}

cleanup:

	printf("%s\n", (errors ? "FAILED" : "Passed"));

	hipFree(A_gpu);
	hipFree(B_gpu);
	free(A_cpu);
	free(B_cpu);

	return 0;
}

