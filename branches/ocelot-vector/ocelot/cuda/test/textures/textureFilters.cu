#include "hip/hip_runtime.h"
/*!
	\file textureFilters.cu

	\author Andrew Kerr <arkerr@gatech.edu>

	\brief tests various filter modes for 2D textures

	\date 27 Oct 2009
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

#define VERBOSE 1

/////////////////////////////////////////////////////////////////////////////////////////////////

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> surface;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelNormCoords(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float sample = tex2D(surface, (float)x / (float)width, (float)y / (float)height);

	out[x + y * width] = sample;
}

static int testNormalizedCoordinates() {
	int width = 64, height = 64;

	float *in_data_host, *out_data_host;
	float *in_data_gpu, *out_data_gpu;

	size_t bytes = width * height * sizeof(float);
	in_data_host = (float *)malloc(bytes);
	out_data_host = (float *)malloc(bytes);

	// procedural texture generation
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = (float)((122 + i*3 + j*2) % 128) / 128.0f;
			out_data_host[i*width+j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMalloc((void **)&in_data_gpu, bytes);
	hipMemcpy(in_data_gpu, in_data_host, bytes, hipMemcpyHostToDevice);

	surface.addressMode[0] = hipAddressModeWrap;
	surface.addressMode[1] = hipAddressModeWrap;
	surface.filterMode = hipFilterModePoint;
	surface.normalized = true;

	if (hipBindTexture2D(0, &surface, in_data_gpu, &channelDesc, width, height, 
		width*sizeof(float)) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytes);

	dim3 grid(width / 16, height / 16), block(16, 16);
	
	kernelNormCoords<<< grid, block >>>(out_data_gpu, width, height);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytes, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	for (int i = 0; i < height && errors < 5; i++) {
		for (int j = 0; j < width && errors < 5; j++) {
			float in = in_data_host[i * width + j];
			float out = out_data_host[i * width + j];
			if (fabs(in - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - in = %f, out = %f %s\n", i, j, in, out, (errors ? "***":""));
			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testNormalizedCoordinates() - failed\n");
	}
#if VERBOSE==1
	else {
		printf("testNormalizedCoordinates() succeeded\n");
	}
#endif

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

texture<ushort, 2, hipReadModeNormalizedFloat> surfaceNormUshort;


/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelNormUshort(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float sample = tex2D(surfaceNormUshort, x, y);

	out[x + y * width] = sample;
}

/*!
	\brief constructs a texture and samples
*/
static int testNormalizedUshort() {
	int width = 128, height = 128;

	ushort *in_data_host, *in_data_gpu;
	float *out_data_host, *out_data_gpu;

	size_t bytesIn = width * height * sizeof(ushort);
	size_t bytesOut = width * height * sizeof(float);
	in_data_host = (ushort *)malloc(bytesIn);
	out_data_host = (float *)malloc(bytesOut);

	// procedural texture generation
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = ((1000 + i*3 + j*2) % (1 << 14));
			out_data_host[i*width+j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, 
		hipChannelFormatKindUnsigned);
	size_t pitch = sizeof(ushort)*width;
	if (hipMallocPitch((void **)&in_data_gpu, &pitch, width * sizeof(ushort), height) 
		!= hipSuccess) {
		printf("hipMallocPitch() failed\n");
		return 1;
	}
	if (hipMemcpy2D(in_data_gpu, pitch, in_data_host, sizeof(ushort)*width, 
		width*sizeof(ushort), height, hipMemcpyHostToDevice) != hipSuccess) {
		printf("hipMemcpy2D() failed\n");
	}

	surfaceNormUshort.addressMode[0] = hipAddressModeWrap;
	surfaceNormUshort.addressMode[1] = hipAddressModeWrap;
	surfaceNormUshort.filterMode = hipFilterModePoint;
	surfaceNormUshort.normalized = false;

	if (hipBindTexture2D(0, &surfaceNormUshort, in_data_gpu, &channelDesc, width, height, 
		pitch) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytesOut);

	dim3 grid(width / 16, height / 16), block(16, 16);
	
	kernelNormUshort<<< grid, block >>>(out_data_gpu, width, height);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytesOut, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	for (int i = 0; i < height && errors < 5; i++) {
		for (int j = 0; j < width && errors < 5; j++) {
			ushort in = in_data_host[i * width + j];
			float out = out_data_host[i * width + j];
			float w = (float)in / (float)(0x0ffff);
			if (fabs(w - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - w = %f, out = %f %s\n", i, j, w, out, (errors ? "***":""));
			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testNormalizedUshort() - failed\n");
	}
#if VERBOSE==1
	else {
		printf("testNormalizedUshort() succeeded\n");
	}
#endif

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

texture<float, 2, hipReadModeElementType> surfaceUpsample;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelUpsample(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float u = (float)x / (float)width;
	float v = (float)y / (float)height;

	float sample = tex2D(surfaceUpsample, u, v);

	out[x + y * width] = sample;
}

/*!
	\brief constructs a texture and samples
*/
static int testUpsample() {
	const int inWidth = 32, inHeight = 32;
	const int outWidth = 64, outHeight = 64;

	float *in_data_host, *in_data_gpu;
	float *out_data_host, *out_data_gpu;

	size_t bytesIn = inWidth * inHeight * sizeof(float);
	size_t bytesOut = outWidth * outHeight * sizeof(float);

	size_t pitch = sizeof(float)*inWidth;
	int errors = 0;

	in_data_host = (float *)malloc(bytesIn);
	out_data_host = (float *)malloc(bytesOut);

	// procedural texture generation
	for (int i = 0; i < inHeight; i++) {
		for (int j = 0; j < inWidth; j++) {
			float x = ((123 + 7 * i + 11 * j) % 1024) / (1024.0f);
			in_data_host[i * inWidth + j] = x;
		}
	}
	for (int i = 0; i < outHeight; i++) {
		for (int j = 0; j < outWidth; j++) {
			out_data_host[i * outWidth + j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, 
		hipChannelFormatKindFloat);

	if (hipMallocPitch((void **)&in_data_gpu, &pitch, inWidth * sizeof(float), inHeight) 
		!= hipSuccess) {
		printf("hipMallocPitch() failed\n");
		return 1;
	}

	if (hipMemcpy(in_data_gpu, in_data_host, bytesIn, hipMemcpyHostToDevice) != hipSuccess) {
		printf("hipMemcpy() failed\n");
	}

	surfaceUpsample.addressMode[0] = hipAddressModeWrap;
	surfaceUpsample.addressMode[1] = hipAddressModeWrap;
	surfaceUpsample.filterMode = hipFilterModePoint;
	surfaceUpsample.normalized = true;

	if (hipBindTexture2D(0, &surfaceUpsample, in_data_gpu, &channelDesc, inWidth, inHeight, 
		pitch) != hipSuccess) {

		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytesOut);
	hipMemcpy(out_data_gpu, out_data_host, bytesOut, hipMemcpyHostToDevice);

	dim3 grid(outWidth / 16, outHeight / 16), block(16, 16);
	
	kernelUpsample<<< grid, block >>>(out_data_gpu, outWidth, outHeight);

	hipDeviceSynchronize();

	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("kernelNormLinear() returned with error %s\n", hipGetErrorString(hipError_t));
		++errors;
	} 

	hipMemcpy(out_data_host, out_data_gpu, bytesOut, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	for (int i = 0; i < outHeight && errors < 5; i++) {
		for (int j = 0; j < outWidth && errors < 5; j++) {
			//
			// simulate nearest point sampling
			//
			float u = (float)j / (float)outWidth, v = (float)i / (float)outHeight;
			
			int tx = (int)(u * (float)inWidth), ty = (int)(v * (float)inHeight);
			float w = in_data_host[tx + inWidth * ty];

			float out = out_data_host[i * outWidth + j];
			if (fabs(w - out) > 0.001f) {
				++errors;
				printf("(row %d, col %d) - w = %f, out = %f %s\n", i, j, w, out, (1 ? " * * * * * * *":""));
				printf("    w = 0x%x\n", *(unsigned int*)&w);
				printf("  out = 0x%x\n", *(unsigned int*)&out);
				printf("    ^ = 0x%x\n", *(unsigned int*)&w ^ *(unsigned int*)&out);
				printf("  (u: %f,  v: %f)\n", u, v);
				printf("  (tx: %d, ty: %d )\n", tx, ty);
			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testUpsample() - failed\n");
	}
#if VERBOSE==1
	else {
		printf("testUpsample() succeeded\n");
	}
#endif

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

texture<float, 2, hipReadModeElementType> surfaceUpsampleLinear;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelUpsampleLinear(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float u = (float)x / (float)width;
	float v = (float)y / (float)height;

	float sample = tex2D(surfaceUpsampleLinear, u, v);

	out[x + y * width] = sample;
}

/*!
	\brief constructs a texture and samples
*/
static int testUpsampleLinear() {
	const int inWidth = 8, inHeight = 8;
	const int outWidth = 16, outHeight = 16;

	float *in_data_host, *in_data_gpu;
	float *out_data_host, *out_data_gpu;

	size_t bytesIn = inWidth * inHeight * sizeof(float);
	size_t bytesOut = outWidth * outHeight * sizeof(float);

	size_t pitch = sizeof(float)*inWidth;
	int errors = 0;

	in_data_host = (float *)malloc(bytesIn);
	out_data_host = (float *)malloc(bytesOut);

	float data[] = {
		0, 0, 0, 0,  0, 0.71f, 0, 0,
		0, 1, 0.5f, 0,  0, 0, 0.531f, 0,
		0, 0.2f, 0, 0,  .934f, 0, .1008, 0,
		0, .11f, .911f, 0,  3.14159, 0, 0, 0,

		0, 0, 0.615f, 0,  0, 0, 0, 0,
		0, 0, 0, 0,  .1205f, 0, 0.23f, 0,
		0, 0.9125f, 0, 0,  0, 0, 0, 0,
		0, 0, 0, 0,  0, 0, 0, 0,
	};

	// procedural texture generation
	int z = 0;
	for (int i = 0; i < inHeight; i++) {
		for (int j = 0; j < inWidth; j++) {
			float x;
			if (z < 64) {
				x = data[z++];
			}
			else {
				x = ((192 + 11 * i + 23 * j) % 1024) / 1024.0f;
			}
			in_data_host[i * inWidth + j] = x;
		}
	}
	for (int i = 0; i < outHeight; i++) {
		for (int j = 0; j < outWidth; j++) {
			out_data_host[i * outWidth + j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, 
		hipChannelFormatKindFloat);
	if (hipMallocPitch((void **)&in_data_gpu, &pitch, inWidth * sizeof(float), inHeight) 
		!= hipSuccess) {
		printf("hipMallocPitch() failed\n");
		return 1;
	}

	if (hipMemcpy(in_data_gpu, in_data_host, bytesIn, hipMemcpyHostToDevice) != hipSuccess) {
		printf("hipMemcpy() failed\n");
	}

	surfaceUpsampleLinear.addressMode[0] = hipAddressModeWrap;
	surfaceUpsampleLinear.addressMode[1] = hipAddressModeWrap;
	surfaceUpsampleLinear.filterMode = hipFilterModeLinear;
	surfaceUpsampleLinear.normalized = true;

	if (hipBindTexture2D(0, &surfaceUpsampleLinear, in_data_gpu, &channelDesc, inWidth, inHeight, 
		pitch) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytesOut);

	dim3 grid(outWidth / 8, outHeight / 8), block(8, 8);
	
	kernelUpsampleLinear<<< grid, block >>>(out_data_gpu, outWidth, outHeight);

	hipDeviceSynchronize();

	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("kernelNormLinear() returned with error %s\n", hipGetErrorString(hipError_t));
		++errors;
	} 

	hipMemcpy(out_data_host, out_data_gpu, bytesOut, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	int fringeV = outHeight / inHeight - 1;
	int fringeH = outWidth / inWidth - 1;
	for (int i = fringeV; i < outHeight - fringeV && errors < 5; i++) {
		for (int j = fringeH; j < outWidth - fringeH && errors < 5; j++) {
			//
			// simulate bilinear sampling
			//
			float u = (float)j / (float)outWidth, v = (float)i / (float)outHeight;
			float ftx = u * (float)inWidth - 0.5f;
			float fty = v * (float)inHeight - 0.5f;
			int tx = (int)ftx, ty = (int)fty;

			float s0 = 0, s1 = 0, s2 = 0, s3 = 0;

			// nearest point sampling of four pixels containing texture sample (u, v)
			s0 = in_data_host[tx + ty * inWidth];
			s1 = in_data_host[tx+1 + ty * inWidth];
			s2 = in_data_host[tx + (ty + 1) * inWidth];
			s3 = in_data_host[tx + 1 + (ty + 1) * inWidth];			

			// bilinear interpolate
			float itu = ftx - (float)tx;
			float itv = fty - (float)ty;

			float w = (s0 * (1.0f - itu) + s1 * itu) * (1.0f - itv) +
				(s2 * (1.0f - itu) + s3 * itu) * itv;

			// correctness test
			float out = out_data_host[i * outWidth + j];
			if (fabs(w - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - w = %f, out = %f %s\n", j, i, w, out, (errors ? "***":""));
				printf("      (u, v) = %f, %f\n", u, v);
				printf("  (itu, itv) = %f, %f\n", itu, itv);
				printf("  s0 = %f\n", s0);
				printf("  s1 = %f\n", s1);
				printf("  s2 = %f\n", s2);
				printf("  s3 = %f\n", s3);

			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testUpsampleLinear() - failed\n");
	}
#if VERBOSE==1
	else {
		printf("testUpsampleLinear() succeeded\n");
	}
#endif


	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **arg) {

	int errors = testNormalizedCoordinates() + testNormalizedUshort() 
		+ testUpsample() + testUpsampleLinear();

	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));
	return (errors ? -1 : 0);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

