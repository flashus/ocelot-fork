
#include <hip/hip_runtime.h>
// file: memoryCheck.cu
__global__ void badMemoryReference(int *A) {
	A[threadIdx.x] = 0;					// line 3 - faulting store
}

int main() {
/*
	int *invalidPtr = 0x0234;		// pointer arbitrarily chosen,
															//   not allocated via cudaMalloc()
*/										
	int *invalidPtr = reinterpret_cast<int *>(0x0234);
	
	int *validPtr = 0;
	hipMalloc((void **)&validPtr, sizeof(int)*64);
	
	badMemoryReference<<< dim3(1,1), dim3(64, 1) >>>( invalidPtr );
	
	return 0;
}

