
#include <hip/hip_runtime.h>
// file: raceCondition.cu
__global__ void raceCondition(int *A) {
	__shared__ int Shared[64];
	
	Shared[threadIdx.x] = A[threadIdx.x];
	
	// no synchronization barrier!
	
	A[threadIdx.x] = Shared[threadIdx.x];	// line 9 - faulting load
}

int main() {
	
	int *validPtr = 0;
	hipMalloc((void **)&validPtr, sizeof(int)*64);
	raceCondition<<< dim3(1,1), dim3(64, 1) >>>( validPtr );
	
	return 0;
}

