/*! \file vectorScale.cu
 *	\author Rodrigo Dominguez <rdomingu@ece.neu.edu>
 *	\date April 18, 2010
 *	\brief Checks basic functionality of the ATI GPU Device
 */

// Standard C Library includes

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void vectorScale(int *A, int *B)
{
	int i = threadIdx.x;
	B[i] = 2 * A[i];
}

int hA[] = {0, 1, 2, 3, 4, 5, 6, 7, 8, 9};
int hB[10];

int main()
{
	int *dA, *dB;
	hipError_t err;

	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, 0);
	if (strcmp(properties.name, "CAL Device")) {
		printf("No CAL devices found\n");
		exit(-1);
	}

	err = hipMalloc((void**)&dA, 10 * sizeof(int));
	if (err != hipSuccess) {
		printf("cudaMalloc() failed: %s\n", hipGetErrorString(err));
		return -1;
	}

	err = hipMalloc((void**)&dB, 10 * sizeof(int));
	if (err != hipSuccess) {
		printf("cudaMalloc() failed: %s\n", hipGetErrorString(err));
		return -1;
	}

	err = hipMemcpy(dA, hA, 10 * sizeof(int), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		printf("cudaMemcpy() failed: %s\n", hipGetErrorString(err));
		return -1;
	}

	vectorScale<<<1, 10>>>(dA, dB);

	err = hipMemcpy(hB, dB, 10 * sizeof(int), hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		printf("cudaMemcpy() failed: %s\n", hipGetErrorString(err));
		return -1;
	}

	err = hipFree(dA);
	if (err != hipSuccess) {
		printf("cudaFree() failed: %s\n", hipGetErrorString(err));
		return -1;
	}

	err = hipFree(dB);
	if (err != hipSuccess) {
		printf("cudaFree() failed: %s\n", hipGetErrorString(err));
		return -1;
	}

	int errors = 0;
	for (int i = 0 ; i < 10 ; i++) {
		int expected = 2 * hA[i];
		int got = hB[i];
		if (abs(expected - got) > 0.001) {
			printf("ERROR - [%d] - got: %d, expected: %d\n", i, got, expected);
			if (++errors > 5) { break; }
		}
	}

	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));

	return 0;
}
