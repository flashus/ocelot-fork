/*!
	\brief loopExit.cu
	\author Andrew Kerr

	\brief simple test of control-flow behavior of kernels
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void kernelLoopExit(int *A, int N) {
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	__shared__ int S[64];
	
	S[threadIdx.x] = 0;
	A[i] = 0;
	for (int j = i; j < N; j++) {
		S[threadIdx.x] += 1;
	}
	__syncthreads();
	A[i] = S[threadIdx.x];
}


int main(int argc, char *arg[]) {

	const int N = 32;
	int *A_host, *A_gpu =0;
	int errors = 0;

	size_t bytes = sizeof(int)*N;

	if (hipMalloc((void **)&A_gpu, bytes) != hipSuccess) {
		printf("cudaMalloc() - failed to allocate %d bytes on device\n", (int)bytes);
		return -1;
	}

	A_host = (int *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		A_host[i] = -1;
	}
	
	hipMemcpy(A_gpu, A_host, bytes, hipMemcpyHostToDevice);

	dim3 grid((N+31)/32,1);
	dim3 block(32, 1);
	
	kernelLoopExit<<< grid, block >>>(A_gpu, N);
	hipMemcpy(A_host, A_gpu, bytes, hipMemcpyDeviceToHost);

	for (int i = 0; (errors < 5) && i < N; ++i) {
		int got = A_host[i];
		int expected = N - i;
		if (expected != got) {
			printf("ERROR 1 [%d] - expected: %d, got: %d\n", i, expected, got);
			++errors;
		}
	}

	hipFree(A_gpu);
	free(A_host);

	if (errors) {
		printf("Pass/Fail : Fail\n");
	}
	else {
		printf("Pass/Fail : Pass\n");
	}

	return 0;
}

