/*!
	\file TestExternalKernel.cu
	\author Andrew Kerr <arkerr@gatech.edu>
	\brief implements tests for external kernel launching
*/


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void testExternalKernel(int *A, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) {
		A[i] = i * 3;		// this gets overridden
	}
}

int main(int argc, char *arg[]) {
	int N = 64;
	size_t bytes = sizeof(int)*N;

	int *A_host, *A_device;	

	A_host = (int *)malloc(bytes);
	hipMalloc((void **)&A_device, bytes);

	testExternalKernel<<< dim3((N+31) / 32, 1), dim3(32, 1) >>>(A_device, N);
	hipMemcpy(A_host, A_device, bytes, hipMemcpyDeviceToHost);

	int errors = 0;

	for (int i = 0; !errors && i < N; i++) {
		if (A_host[i] != i * 4) {
			++errors;
		}
	}

	free(A_host);
	hipFree(A_device);

	if (errors) {
		printf("Test FAILED\n");
	}
	else {
		printf("Test PASSED\n");
	}

	return 0;
}

