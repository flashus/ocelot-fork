#include "hip/hip_runtime.h"
/*!
	\file indirectCall.cu
	
	\author Andrew Kerr <arkerr@gatech.edu>
	
	\brief demonstrates indirect function calling
*/

#include <iostream>

extern "C" __device__ __noinline__ int funcDouble(int a) {
	return a*2;
}

extern "C" __device__ __noinline__ int funcTriple(int a) {
	return a*3;
}

extern "C" __device__ __noinline__ int funcQuadruple(int a) {
	return a*4;
}

extern "C" __global__ void kernelEntry(int *A, const int N, int b) {

	int (*filter[])(int) = {
		&funcDouble,
		&funcTriple,
		&funcQuadruple
	};

	int i = threadIdx.x + blockDim.x * blockIdx.x;
	
	if (i < N) {
		int p = ((b + i) % 3);
		A[i] = filter[p](i);
	}
}

int main(int argc, char *arg[]) {

	const int P = 0;
	const int N = 32;
	int *A_gpu, *A_cpu;
	size_t bytes = sizeof(int) * N;
	
	hipError_t result = hipDeviceSynchronize();
	if (result != hipSuccess) {
		std::cout << "Initialization error: " << hipGetErrorString(result) << std::endl;
		return 1;
	}
	
	result = hipMalloc((void **)&A_gpu, bytes);
	if (result != hipSuccess) {
		std::cout << "hipMalloc() - failed to allocate " << bytes << " on the device" << std::endl;
		return 2;
	}
	
	A_cpu = (int *)malloc(bytes);
	for (int i = 0; i < N; i++) {
		A_cpu[i] = 0;
	}
	
	result = hipMemcpy(A_gpu, A_cpu, bytes, hipMemcpyHostToDevice);
	if (result != hipSuccess) {
		std::cout << "hipMemcpy() - failed to copy " << bytes << " bytes TO the device" << std::endl;
		return 2;
	}
	
	dim3 block(32, 1);
	dim3 grid((N + block.x - 1) / block.x, 1);
	
	kernelEntry<<< grid, block >>>(A_gpu, N, P);
	
	result = hipDeviceSynchronize();
	if (result != hipSuccess) {
		std::cout << "Kernel launch error: " << hipGetErrorString(result) << std::endl;
		return 3;
	}
	
	result = hipMemcpy(A_cpu, A_gpu, bytes, hipMemcpyDeviceToHost);
	if (result != hipSuccess) {
		std::cout << "hipMemcpy() - failed to copy " << bytes << " bytes FROM the device" << std::endl;
		return 2;
	}
	
	int errors = 0;
	for (int i = 0; i < N; i++) {
		int got = A_cpu[i];
		int dem = 0;
		
		int p = ((P + i) % 3);
		dem = ((p + 2) * i);
		
		if (got != dem) {
			std::cout << "Error[" << i << "] - expected: " << dem << ", got: " << got << std::endl;
			if (++errors > 5) {
				break;
			}
		}
	}
	
	hipFree(A_gpu);
	free(A_cpu);
	
	if (errors) {
		std::cout << "FAILED" << std::endl;
		std::cout << " with " << errors << " errors" << std::endl;
	}
	else {
		std::cout << "Pass" << std::endl;
	}
	
	return 0;
}

