#include "hip/hip_runtime.h"
/*!
	\file textureFilters.cu

	\author Andrew Kerr <arkerr@gatech.edu>

	\brief tests various filter modes for 2D textures

	\date 27 Oct 2009
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> surface;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelNormCoords(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float sample = tex2D(surface, (float)x / (float)width, (float)y / (float)height);

	out[x + y * width] = sample;
}

static int testNormalizedCoordinates() {
	int width = 64, height = 64;

	float *in_data_host, *out_data_host;
	float *in_data_gpu, *out_data_gpu;

	size_t bytes = width * height * sizeof(float);
	in_data_host = (float *)malloc(bytes);
	out_data_host = (float *)malloc(bytes);

	// procedural texture generation
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = (float)((122 + i*3 + j*2) % 128) / 128.0f;
			out_data_host[i*width+j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMalloc((void **)&in_data_gpu, bytes);
	hipMemcpy(in_data_gpu, in_data_host, bytes, hipMemcpyHostToDevice);

	surface.addressMode[0] = hipAddressModeWrap;
	surface.addressMode[1] = hipAddressModeWrap;
	surface.filterMode = hipFilterModePoint;
	surface.normalized = true;

	if (hipBindTexture2D(0, &surface, in_data_gpu, &channelDesc, width, height, 
		width*sizeof(float)) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytes);

	dim3 grid(width / 16, height / 16), block(16, 16);
	
	kernelNormCoords<<< grid, block >>>(out_data_gpu, width, height);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytes, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	for (int i = 0; i < height && errors < 5; i++) {
		for (int j = 0; j < width && errors < 5; j++) {
			float in = in_data_host[i * width + j];
			float out = out_data_host[i * width + j];
			if (fabs(in - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - in = %f, out = %f %s\n", i, j, in, out, (errors ? "***":""));
			}
		}
	}

	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

texture<ushort, 2, hipReadModeNormalizedFloat> surfaceNormUshort;


/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelNormUshort(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float sample = tex2D(surfaceNormUshort, x, y);

	out[x + y * width] = sample;
}

/*!
	\brief constructs a texture and samples
*/
static int testNormalizedUshort() {
	int width = 64, height = 64;

	ushort *in_data_host, *in_data_gpu;
	float *out_data_host, *out_data_gpu;

	size_t bytesIn = width * height * sizeof(ushort);
	size_t bytesOut = width * height * sizeof(float);
	in_data_host = (ushort *)malloc(bytesIn);
	out_data_host = (float *)malloc(bytesOut);

	// procedural texture generation
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = ((10000 + i*3 + j*2) % (1 << 15));
			out_data_host[i*width+j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipMalloc((void **)&in_data_gpu, bytesIn);
	hipMemcpy(in_data_gpu, in_data_host, bytesIn, hipMemcpyHostToDevice);

	surfaceNormUshort.addressMode[0] = hipAddressModeWrap;
	surfaceNormUshort.addressMode[1] = hipAddressModeWrap;
	surfaceNormUshort.filterMode = hipFilterModePoint;
	surfaceNormUshort.normalized = false;

	if (hipBindTexture2D(0, &surfaceNormUshort, in_data_gpu, &channelDesc, width, height, 
		width*sizeof(float)) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytesOut);

	dim3 grid(width / 16, height / 16), block(16, 16);
	
	kernelNormUshort<<< grid, block >>>(out_data_gpu, width, height);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytesOut, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	for (int i = 0; i < height && errors < 5; i++) {
		for (int j = 0; j < width && errors < 5; j++) {
			ushort in = in_data_host[i * width + j];
			float out = out_data_host[i * width + j];
			float w = (float)in / (float)(0x0ffff);
			if (fabs(w - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - w = %f, out = %f %s\n", i, j, w, out, (errors ? "***":""));
			}
		}
	}

	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **arg) {
	int errors = testNormalizedCoordinates() + testNormalizedUshort();

	return (errors ? -1 : 0);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

