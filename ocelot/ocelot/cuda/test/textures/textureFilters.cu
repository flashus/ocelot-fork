#include "hip/hip_runtime.h"
/*!
	\file textureFilters.cu

	\author Andrew Kerr <arkerr@gatech.edu>

	\brief tests various filter modes for 2D textures

	\date 27 Oct 2009
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

/////////////////////////////////////////////////////////////////////////////////////////////////

// declare texture reference for 2D float texture
texture<float, 2, hipReadModeElementType> surface;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelNormCoords(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float sample = tex2D(surface, (float)x / (float)width, (float)y / (float)height);

	out[x + y * width] = sample;
}

static int testNormalizedCoordinates() {
	int width = 64, height = 64;

	float *in_data_host, *out_data_host;
	float *in_data_gpu, *out_data_gpu;

	size_t bytes = width * height * sizeof(float);
	in_data_host = (float *)malloc(bytes);
	out_data_host = (float *)malloc(bytes);

	// procedural texture generation
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = (float)((122 + i*3 + j*2) % 128) / 128.0f;
			out_data_host[i*width+j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipMalloc((void **)&in_data_gpu, bytes);
	hipMemcpy(in_data_gpu, in_data_host, bytes, hipMemcpyHostToDevice);

	surface.addressMode[0] = hipAddressModeWrap;
	surface.addressMode[1] = hipAddressModeWrap;
	surface.filterMode = hipFilterModePoint;
	surface.normalized = true;

	if (hipBindTexture2D(0, &surface, in_data_gpu, &channelDesc, width, height, 
		width*sizeof(float)) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytes);

	dim3 grid(width / 16, height / 16), block(16, 16);
	
	kernelNormCoords<<< grid, block >>>(out_data_gpu, width, height);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytes, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	for (int i = 0; i < height && errors < 5; i++) {
		for (int j = 0; j < width && errors < 5; j++) {
			float in = in_data_host[i * width + j];
			float out = out_data_host[i * width + j];
			if (fabs(in - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - in = %f, out = %f %s\n", i, j, in, out, (errors ? "***":""));
			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testNormalizedCoordinates() - failed\n");
	}

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

texture<ushort, 2, hipReadModeNormalizedFloat> surfaceNormUshort;


/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelNormUshort(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float sample = tex2D(surfaceNormUshort, x, y);

	out[x + y * width] = sample;
}

/*!
	\brief constructs a texture and samples
*/
static int testNormalizedUshort() {
	int width = 128, height = 128;

	ushort *in_data_host, *in_data_gpu;
	float *out_data_host, *out_data_gpu;

	size_t bytesIn = width * height * sizeof(ushort);
	size_t bytesOut = width * height * sizeof(float);
	in_data_host = (ushort *)malloc(bytesIn);
	out_data_host = (float *)malloc(bytesOut);

	// procedural texture generation
	for (int i = 0; i < height; i++) {
		for (int j = 0; j < width; j++) {
			in_data_host[i * width + j] = ((1000 + i*3 + j*2) % (1 << 14));
			out_data_host[i*width+j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(16, 0, 0, 0, 
		hipChannelFormatKindUnsigned);
	size_t pitch = sizeof(ushort)*width;
	if (hipMallocPitch((void **)&in_data_gpu, &pitch, width * sizeof(ushort), height) 
		!= hipSuccess) {
		printf("hipMallocPitch() failed\n");
		return 1;
	}
	if (hipMemcpy2D(in_data_gpu, pitch, in_data_host, sizeof(ushort)*width, 
		width*sizeof(ushort), height, hipMemcpyHostToDevice) != hipSuccess) {
		printf("hipMemcpy2D() failed\n");
	}

	surfaceNormUshort.addressMode[0] = hipAddressModeWrap;
	surfaceNormUshort.addressMode[1] = hipAddressModeWrap;
	surfaceNormUshort.filterMode = hipFilterModePoint;
	surfaceNormUshort.normalized = false;

	if (hipBindTexture2D(0, &surfaceNormUshort, in_data_gpu, &channelDesc, width, height, 
		pitch) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytesOut);

	dim3 grid(width / 16, height / 16), block(16, 16);
	
	kernelNormUshort<<< grid, block >>>(out_data_gpu, width, height);

	hipDeviceSynchronize();

	hipMemcpy(out_data_host, out_data_gpu, bytesOut, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	int errors = 0;
	for (int i = 0; i < height && errors < 5; i++) {
		for (int j = 0; j < width && errors < 5; j++) {
			ushort in = in_data_host[i * width + j];
			float out = out_data_host[i * width + j];
			float w = (float)in / (float)(0x0ffff);
			if (fabs(w - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - w = %f, out = %f %s\n", i, j, w, out, (errors ? "***":""));
			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testNormalizedUshort() - failed\n");
	}

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

texture<float, 2, hipReadModeElementType> surfaceUpsample;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelUpsample(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float u = (float)x / (float)width;
	float v = (float)y / (float)height;

	float sample = tex2D(surfaceUpsample, u, v);

	out[x + y * width] = sample;
}

/*!
	\brief constructs a texture and samples
*/
static int testUpsample() {
	const int inWidth = 32, inHeight = 32;
	const int outWidth = 128, outHeight = 128;

	float *in_data_host, *in_data_gpu;
	float *out_data_host, *out_data_gpu;

	size_t bytesIn = inWidth * inHeight * sizeof(float);
	size_t bytesOut = outWidth * outHeight * sizeof(float);

	size_t pitch = sizeof(float)*inWidth;
	int errors = 0;

	in_data_host = (float *)malloc(bytesIn);
	out_data_host = (float *)malloc(bytesOut);

	// procedural texture generation
	for (int i = 0; i < inHeight; i++) {
		for (int j = 0; j < inWidth; j++) {
			float x = ((123 + 7 * i + 11 * j) % 1024) / (1024.0f);
			in_data_host[i * inWidth + j] = x;
		}
	}
	for (int i = 0; i < outHeight; i++) {
		for (int j = 0; j < outWidth; j++) {
			out_data_host[i * outWidth + j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, 
		hipChannelFormatKindFloat);
	if (hipMallocPitch((void **)&in_data_gpu, &pitch, inWidth * sizeof(float), inHeight) 
		!= hipSuccess) {
		printf("hipMallocPitch() failed\n");
		return 1;
	}
	if (hipMemcpy2D(in_data_gpu, pitch, in_data_host, sizeof(float)*inWidth, 
		inWidth*sizeof(float), inHeight, hipMemcpyHostToDevice) != hipSuccess) {
		printf("hipMemcpy2D() failed\n");
	}

	surfaceUpsample.addressMode[0] = hipAddressModeWrap;
	surfaceUpsample.addressMode[1] = hipAddressModeWrap;
	surfaceUpsample.filterMode = hipFilterModePoint;
	surfaceUpsample.normalized = true;

	if (hipBindTexture2D(0, &surfaceUpsample, in_data_gpu, &channelDesc, inWidth, inHeight, 
		pitch) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytesOut);

	dim3 grid(outWidth / 16, outHeight / 16), block(16, 16);
	
	kernelUpsample<<< grid, block >>>(out_data_gpu, outWidth, outHeight);

	hipDeviceSynchronize();

	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("kernelNormLinear() returned with error %s\n", hipGetErrorString(hipError_t));
		++errors;
	} 

	hipMemcpy(out_data_host, out_data_gpu, bytesOut, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	for (int i = 0; i < outHeight && errors < 5; i++) {
		for (int j = 0; j < outWidth && errors < 5; j++) {
			//
			// simulate nearest point sampling
			//
			float u = (float)j / (float)outWidth, v = (float)i / (float)outHeight;
			
			int tx = (int)(u * (float)inWidth), ty = (int)(v * (float)inHeight);
			float w = in_data_host[tx + inWidth * ty];

			float out = out_data_host[i * outWidth + j];
			if (fabs(w - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - w = %f, out = %f %s\n", i, j, w, out, (errors ? "***":""));
			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testUpsample() - failed\n");
	}

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////

texture<float, 2, hipReadModeElementType> surfaceUpsampleLinear;

/*!
	kernel in which each thread samples the texture and writes it to out, a row-major dense 
	block of samples
*/
extern "C" __global__ void kernelUpsampleLinear(float *out, int width, int height) {
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float u = (float)x / (float)width;
	float v = (float)y / (float)height;

	float sample = tex2D(surfaceUpsampleLinear, u, v);

	out[x + y * width] = sample;
}

/*!
	\brief constructs a texture and samples
*/
static int testUpsampleLinear() {
	const int inWidth = 32, inHeight = 32;
	const int outWidth = 128, outHeight = 128;

	float *in_data_host, *in_data_gpu;
	float *out_data_host, *out_data_gpu;

	size_t bytesIn = inWidth * inHeight * sizeof(float);
	size_t bytesOut = outWidth * outHeight * sizeof(float);

	size_t pitch = sizeof(float)*inWidth;
	int errors = 0;

	in_data_host = (float *)malloc(bytesIn);
	out_data_host = (float *)malloc(bytesOut);

	// procedural texture generation
	for (int i = 0; i < inHeight; i++) {
		for (int j = 0; j < inWidth; j++) {
			float x = ((123 + 7 * i + 11 * j) % 1024) / (1024.0f);
			in_data_host[i * inWidth + j] = x;
		}
	}
	for (int i = 0; i < outHeight; i++) {
		for (int j = 0; j < outWidth; j++) {
			out_data_host[i * outWidth + j] = 0;
		}
	}

	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, 
		hipChannelFormatKindFloat);
	if (hipMallocPitch((void **)&in_data_gpu, &pitch, inWidth * sizeof(float), inHeight) 
		!= hipSuccess) {
		printf("hipMallocPitch() failed\n");
		return 1;
	}
	if (hipMemcpy2D(in_data_gpu, pitch, in_data_host, sizeof(float)*inWidth, 
		inWidth*sizeof(float), inHeight, hipMemcpyHostToDevice) != hipSuccess) {
		printf("hipMemcpy2D() failed\n");
	}

	surfaceUpsampleLinear.addressMode[0] = hipAddressModeWrap;
	surfaceUpsampleLinear.addressMode[1] = hipAddressModeWrap;
	surfaceUpsampleLinear.filterMode = hipFilterModeLinear;
	surfaceUpsampleLinear.normalized = true;

	if (hipBindTexture2D(0, &surfaceUpsampleLinear, in_data_gpu, &channelDesc, inWidth, inHeight, 
		pitch) != hipSuccess) {
		printf("failed to bind texture: %s\n", hipGetErrorString(hipGetLastError()));
		free(in_data_host);
		free(out_data_host);
		hipFree(in_data_gpu);
		return -2;
	}

	hipMalloc((void **)&out_data_gpu, bytesOut);

	dim3 grid(outWidth / 16, outHeight / 16), block(16, 16);
	
	kernelUpsampleLinear<<< grid, block >>>(out_data_gpu, outWidth, outHeight);

	hipDeviceSynchronize();

	hipError_t hipError_t = hipGetLastError();
	if (hipError_t != hipSuccess) {
		printf("kernelNormLinear() returned with error %s\n", hipGetErrorString(hipError_t));
		++errors;
	} 

	hipMemcpy(out_data_host, out_data_gpu, bytesOut, hipMemcpyDeviceToHost);
	hipFree(in_data_gpu);
	hipFree(out_data_gpu);

	printf("\n\nChecking bilinear interpolation\n");

	for (int i = 4; i < outHeight - 4 && errors < 5; i++) {
		for (int j = 4; j < outWidth - 4 && errors < 5; j++) {
			//
			// simulate bilinear sampling
			//
			float u = (float)j / (float)outWidth, v = (float)i / (float)outHeight;
			int tx = (int)(u * (float)inWidth), ty = (int)(v * (float)inHeight);

			float s0 = 0, s1 = 0, s2 = 0, s3 = 0;

			// nearest point sampling of four pixels containing texture sample (u, v)
			s0 = in_data_host[tx + ty * inWidth];
			s1 = in_data_host[tx+1 + ty * inWidth];
			s2 = in_data_host[tx + (ty + 1) * inWidth];
			s3 = in_data_host[tx + 1 + (ty + 1) * inWidth];			

			// bilinear interpolate
			float itu = (u * (float)inWidth) - (float)tx;
			float itv = (v * (float)inHeight) - (float)ty;

			float w = (s0 * (1.0f - itu) + s1 * itu) * (1.0f - itv) +
				(s2 * (1.0f - itu) + s3 * itu) * itv;
			
			// correctness test
			float out = out_data_host[i * outWidth + j];
			if (fabs(w - out) > 0.001f) {
				++errors;
				printf("(%d, %d) - w = %f, out = %f %s\n", j, i, w, out, (errors ? "***":""));
				printf("      (u, v) = %f, %f\n", u, v);
				printf("  (itu, itv) = %f, %f\n", itu, itv);
				printf("  s0 = %f\n", s0);
				printf("  s1 = %f\n", s1);
				printf("  s2 = %f\n", s2);
				printf("  s3 = %f\n", s3);
			}
		}
	}

	if (errors) {
		printf("FAILED\n\n testUpsampleLinear() - failed\n");
	}

	free(in_data_host);
	free(out_data_host);
	return errors;
}

/////////////////////////////////////////////////////////////////////////////////////////////////

int main(int argc, char **arg) {
	int errors = testNormalizedCoordinates() + testNormalizedUshort() 
		+ testUpsample();

	printf("Pass/Fail : %s\n", (errors ? "Fail" : "Pass"));
	return (errors ? -1 : 0);
}

/////////////////////////////////////////////////////////////////////////////////////////////////

